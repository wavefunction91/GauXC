#include "hip/hip_runtime.h"
#include "cuda_eval_denvars.hpp"
#include "cuda_extensions.hpp"
#include <gauxc/util/div_ceil.hpp>

#include "cuda_device_properties.hpp"

namespace GauXC      {
namespace integrator {
namespace cuda       {

using namespace GauXC::cuda;

template <typename T>
__global__ void eval_uvars_lda_kernel( size_t           ntasks,
                                       XCTaskDevice<T>* tasks_device ) {

  const int batch_idx = blockIdx.z;
  if( batch_idx >= ntasks ) return;

  auto& task = tasks_device[ batch_idx ];

  const auto npts            = task.npts;
  const auto nbf             = task.nbe;

  auto* den_eval_device   = task.den;

  const auto* basis_eval_device = task.bf;

  const auto* den_basis_prod_device = task.zmat;

  const int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  const int tid_y = blockIdx.y * blockDim.y + threadIdx.y;

  register double den_reg = 0.;

  if( tid_x < nbf and tid_y < npts ) {

    const double* bf_col   = basis_eval_device     + tid_y*nbf;
    const double* db_col   = den_basis_prod_device + tid_y*nbf;

    den_reg = bf_col[ tid_x ]   * db_col[ tid_x ];

  }

  // Warp blocks are stored col major
  den_reg = 2 * warpReduceSum( den_reg );


  if( threadIdx.x == 0 and tid_y < npts ) {
    atomicAdd( den_eval_device   + tid_y, den_reg );
  }
  

}

template <typename T>
__global__ void eval_uvars_gga_kernel( size_t           ntasks,
                                       XCTaskDevice<T>* tasks_device ) {

  const int batch_idx = blockIdx.z;
  if( batch_idx >= ntasks ) return;

  auto& task = tasks_device[ batch_idx ];

  const auto npts            = task.npts;
  const auto nbf             = task.nbe;

  auto* den_eval_device   = task.den;
  auto* den_x_eval_device = task.ddenx;
  auto* den_y_eval_device = task.ddeny;
  auto* den_z_eval_device = task.ddenz;

  const auto* basis_eval_device = task.bf;
  const auto* dbasis_x_eval_device = task.dbfx;
  const auto* dbasis_y_eval_device = task.dbfy;
  const auto* dbasis_z_eval_device = task.dbfz;

  const auto* den_basis_prod_device = task.zmat;

  const int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  const int tid_y = blockIdx.y * blockDim.y + threadIdx.y;

  register double den_reg = 0.;
  register double dx_reg  = 0.;
  register double dy_reg  = 0.;
  register double dz_reg  = 0.;

  if( tid_x < nbf and tid_y < npts ) {

    const double* bf_col   = basis_eval_device     + tid_y*nbf;
    const double* bf_x_col = dbasis_x_eval_device  + tid_y*nbf;
    const double* bf_y_col = dbasis_y_eval_device  + tid_y*nbf;
    const double* bf_z_col = dbasis_z_eval_device  + tid_y*nbf;
    const double* db_col   = den_basis_prod_device + tid_y*nbf;

    den_reg = bf_col[ tid_x ]   * db_col[ tid_x ];
    dx_reg  = bf_x_col[ tid_x ] * db_col[ tid_x ];
    dy_reg  = bf_y_col[ tid_x ] * db_col[ tid_x ];
    dz_reg  = bf_z_col[ tid_x ] * db_col[ tid_x ];

  }

  // Warp blocks are stored col major
  den_reg = 2 * warpReduceSum( den_reg );
  dx_reg  = 4 * warpReduceSum( dx_reg );
  dy_reg  = 4 * warpReduceSum( dy_reg );
  dz_reg  = 4 * warpReduceSum( dz_reg );


  if( threadIdx.x == 0 and tid_y < npts ) {
    atomicAdd( den_eval_device   + tid_y, den_reg );
    atomicAdd( den_x_eval_device + tid_y, dx_reg  );
    atomicAdd( den_y_eval_device + tid_y, dy_reg  );
    atomicAdd( den_z_eval_device + tid_y, dz_reg  );
  }
  

}


template <typename T>
__global__ void eval_vvars_gga_kernel( 
  size_t   npts,
  const T* den_x_eval_device,
  const T* den_y_eval_device,
  const T* den_z_eval_device,
        T* gamma_eval_device
) {

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if( tid < npts ) {

    const double dx = den_x_eval_device[ tid ];
    const double dy = den_y_eval_device[ tid ];
    const double dz = den_z_eval_device[ tid ];

    gamma_eval_device[tid] = dx*dx + dy*dy + dz*dz;

  }

}




template <typename T>
void eval_uvars_lda_device( size_t           ntasks,
                            size_t           max_nbf,
                            size_t           max_npts,
                            XCTaskDevice<T>* tasks_device,
                            hipStream_t     stream ) {

  dim3 threads(warp_size, max_warps_per_thread_block, 1);
  dim3 blocks( util::div_ceil( max_nbf , threads.x ),
               util::div_ceil( max_npts , threads.y ),
               ntasks );

  eval_uvars_lda_kernel<<< blocks, threads, 0, stream >>>( ntasks, tasks_device );

}

template <typename T>
void eval_uvars_gga_device( size_t           ntasks,
                            size_t           max_nbf,
                            size_t           max_npts,
                            XCTaskDevice<T>* tasks_device,
                            hipStream_t     stream ) {

  dim3 threads(warp_size, max_warps_per_thread_block, 1);
  dim3 blocks( util::div_ceil( max_nbf , threads.x ),
               util::div_ceil( max_npts , threads.y ),
               ntasks );

  eval_uvars_gga_kernel<<< blocks, threads, 0, stream >>>( ntasks, tasks_device );

}
 

template <typename T>
void eval_vvars_gga_device( size_t       npts,
                            const T*     den_x_device,
                            const T*     den_y_device,
                            const T*     den_z_device,
                                  T*     gamma_device,
                            hipStream_t stream ) {

  dim3 threads( max_threads_per_thread_block );
  dim3 blocks( util::div_ceil( npts, threads.x ) );

  eval_vvars_gga_kernel<<< blocks, threads, 0, stream >>>(
    npts, den_x_device, den_y_device, den_z_device, gamma_device
  );

}
                          














template
void eval_uvars_lda_device( size_t                ntasks,
                            size_t                max_nbf,
                            size_t                max_npts,
                            XCTaskDevice<double>* tasks_device,
                            hipStream_t          stream );

template
void eval_uvars_gga_device( size_t                ntasks,
                            size_t                max_nbf,
                            size_t                max_npts,
                            XCTaskDevice<double>* tasks_device,
                            hipStream_t          stream );

template
void eval_vvars_gga_device( size_t            npts,
                            const double*     den_x_device,
                            const double*     den_y_device,
                            const double*     den_z_device,
                                  double*     gamma_device,
                            hipStream_t      stream );

}
}
}
