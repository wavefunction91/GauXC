#include "hip/hip_runtime.h"
/**
 * GauXC Copyright (c) 2020-2024, The Regents of the University of California,
 * through Lawrence Berkeley National Laboratory (subject to receipt of
 * any required approvals from the U.S. Dept. of Energy). All rights reserved.
 *
 * See LICENSE.txt for details
 */
#include <gauxc/util/div_ceil.hpp>
#include "exceptions/cublas_exception.hpp"

#include "device_specific/cuda_device_constants.hpp"
#include "device_specific/cublas_util.hpp"
#include "device/common/device_blas.hpp"

namespace GauXC {

hipblasOperation_t device_op_to_cublas( DeviceBlasOp op ) {
  switch( op ) {
    case DeviceBlasOp::NoTrans: return HIPBLAS_OP_N;
    case DeviceBlasOp::Trans:   return HIPBLAS_OP_T;
    default:
      GAUXC_GENERIC_EXCEPTION("Unsupported DeviceBlasOp");
      return HIPBLAS_OP_N;
  }
}

hipblasFillMode_t device_uplo_to_cublas( DeviceBlasUplo uplo ) {
  switch(uplo) {
    case DeviceBlasUplo::Upper: return HIPBLAS_FILL_MODE_UPPER;
    case DeviceBlasUplo::Lower: return HIPBLAS_FILL_MODE_LOWER;
    default:
      GAUXC_GENERIC_EXCEPTION("Unsupported DeviceBlasUplo");
      return HIPBLAS_FILL_MODE_LOWER;
  }
}

template <typename T>
__global__ void increment_kernel( const T* X, T* Y ) {
  const auto tid = blockIdx.x;
  if( tid < 1 ) (*Y) += (*X);
}

template <typename T>
void increment( const T* X, T* Y, hipStream_t stream ) {
  increment_kernel<<<1,1,0,stream>>>(X,Y);
}

template <>
void dot( device_blas_handle generic_handle,
          int            N,
          const double*  X,
          int            INCX,
          const double*  Y,
          int            INCY,
          double*        RES ) {

  hipblasHandle_t handle = generic_handle.blas_handle_as<util::cublas_handle>();

  auto stat = hipblasDdot( handle, N, X, INCX, Y, INCY, RES );
  GAUXC_CUBLAS_ERROR("CUBLAS DDOT FAILED", stat );

}

template <typename T>
void gdot( device_blas_handle generic_handle,
           int       N,
           const T*  X,
           int       INCX,
           const T*  Y,
           int       INCY,
           T*        SCR,
           T*        RES ) {


  dot( generic_handle, N, X, INCX, Y, INCY, SCR );
  hipblasHandle_t handle = generic_handle.blas_handle_as<util::cublas_handle>();
  auto stream = util::get_stream(handle);
  increment( SCR, RES, stream );

}

template 
void gdot( device_blas_handle generic_handle,
           int            N,
           const double*  X,
           int            INCX,
           const double*  Y,
           int            INCY,
           double*        SCR,
           double*        RES );










template <typename T>
void __global__ hadamard_product_kernel( int      M,
                                         int      N,
                                         const T* A,
                                         int      LDA,
                                         T*       B,
                                         int      LDB ) {

  const int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  const int tid_y = blockIdx.y * blockDim.y + threadIdx.y;

  if( tid_x < M and tid_y < N ) {
    B[ tid_x + tid_y*LDB ] *= A[ tid_x + tid_y*LDA ];
  }

}



template <typename T>
void hadamard_product( device_blas_handle generic_handle,
                       int            M,
                       int            N,
                       const T*       A,
                       int            LDA,
                       T*             B,
                       int            LDB ) {


  hipblasHandle_t handle = generic_handle.blas_handle_as<util::cublas_handle>();
  auto stream = util::get_stream(handle);
  dim3 threads(cuda::warp_size, cuda::max_warps_per_thread_block);
  dim3 blocks( util::div_ceil( M, threads.x ),
               util::div_ceil( N, threads.y ) );

  hadamard_product_kernel<<< blocks, threads, 0, stream >>>( M, N, A, LDA, B, LDB );

}
 
template 
void hadamard_product( device_blas_handle generic_handle,
                       int            M,
                       int            N,
                       const double*  A,
                       int            LDA,
                       double*        B,
                       int            LDB ); 




template <>
void gemm( device_blas_handle generic_handle, 
           DeviceBlasOp TA, DeviceBlasOp TB,
           int M, int N, int K, double ALPHA, 
           const double* A, int LDA, const double* B, int LDB,
           double BETA, double* C, int LDC ) {


  hipblasHandle_t handle = generic_handle.blas_handle_as<util::cublas_handle>();
  auto stat = hipblasDgemm( handle, device_op_to_cublas(TA), 
    device_op_to_cublas(TB), M, N, K, &ALPHA, A, LDA,
    B, LDB, &BETA, C, LDC );
  GAUXC_CUBLAS_ERROR("CUBLAS DGEMM FAILED", stat);

}


template <>
void syr2k( device_blas_handle generic_handle, 
            DeviceBlasUplo UPLO, DeviceBlasOp Trans,
            int M, int K, double ALPHA, 
            const double* A, int LDA, const double* B, int LDB,
            double BETA, double* C, int LDC ) {

  hipblasHandle_t handle = generic_handle.blas_handle_as<util::cublas_handle>();
  auto stat = hipblasDsyr2k( handle, device_uplo_to_cublas(UPLO), 
    device_op_to_cublas(Trans), M, K, &ALPHA, A, LDA, B, LDB,
    &BETA, C, LDC );
  GAUXC_CUBLAS_ERROR("CUBLAS DSYR2K FAILED", stat);

}

}

