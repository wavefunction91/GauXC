#include "hip/hip_runtime.h"
/**
 * GauXC Copyright (c) 2020-2024, The Regents of the University of California,
 * through Lawrence Berkeley National Laboratory (subject to receipt of
 * any required approvals from the U.S. Dept. of Energy). All rights reserved.
 *
 * See LICENSE.txt for details
 */
#include "device/common/uvvars.hpp"
#include "cuda_extensions.hpp"
#include "device_specific/cuda_device_constants.hpp"
#include <gauxc/util/div_ceil.hpp>
#include "device_specific/cuda_util.hpp"
#include "device/xc_device_data.hpp"

namespace GauXC {

#define VVAR_KERNEL_SM_BLOCK 32
#define GGA_KERNEL_SM_WARPS 16
#define MGGA_KERNEL_SM_BLOCK 32

__global__ void eval_uvars_lda_rks_kernel( size_t ntasks, XCDeviceTask* tasks_device) {
  // eval_vvars populated uvar storage already in the case of LDA+RKS
  return;
}

__global__ void eval_uvars_lda_uks_kernel( size_t        ntasks,
                                       XCDeviceTask* tasks_device ) {

  const int batch_idx = blockIdx.z;
  if( batch_idx >= ntasks ) return;

  auto& task = tasks_device[ batch_idx ];

  const auto npts            = task.npts;

  auto* den_pos_eval_device   = task.den_s;
  auto* den_neg_eval_device   = task.den_z;


  const int tid = blockIdx.x * blockDim.x + threadIdx.x;


  if( tid < npts ) {
    const auto ps = den_pos_eval_device[ tid ];
    const auto pz = den_neg_eval_device[ tid ];
    den_pos_eval_device[ tid ] = 0.5*(ps + pz);
    den_neg_eval_device[ tid ] = 0.5*(ps - pz);

  }
}

__global__ void eval_uvars_lda_gks_kernel( size_t        ntasks,
                                       XCDeviceTask* tasks_device ) {

  const int batch_idx = blockIdx.z;
  if( batch_idx >= ntasks ) return;

  auto& task = tasks_device[ batch_idx ];

  const auto npts            = task.npts;

  auto* den_z_eval_device   = task.den_s;
  auto* den_s_eval_device   = task.den_z;
  auto* den_y_eval_device   = task.den_y;
  auto* den_x_eval_device   = task.den_x;
  auto* K_z_eval_device     = task.K_z;
  auto* K_y_eval_device     = task.K_y;
  auto* K_x_eval_device     = task.K_x;
  const double dtolsq = 1e-24;  // TODO: make variable

  const int tid = blockIdx.x * blockDim.x + threadIdx.x;


  if( tid < npts ) {
    const auto ps = den_s_eval_device[ tid ];
    const auto pz = den_z_eval_device[ tid ];
    const auto py = den_y_eval_device[ tid ];
    const auto px = den_x_eval_device[ tid ];
    const auto mtemp = pz*pz + px*px + py*py;
    double mnorm = 0.;
  
    if (mtemp > dtolsq) {
      const double inv_mnorm = rsqrt(mtemp);
      mnorm = 1./inv_mnorm;
      K_z_eval_device[ tid ] = pz * inv_mnorm;
      K_y_eval_device[ tid ] = py * inv_mnorm;
      K_x_eval_device[ tid ] = px * inv_mnorm;
    }
    else {
      mnorm = (1. / 3.) * (px + py + pz);
      K_z_eval_device[ tid ] = 1. / 3.;
      K_y_eval_device[ tid ] = 1. / 3.;
      K_x_eval_device[ tid ] = 1. / 3.;
    }

    den_s_eval_device[ tid ] = 0.5*(ps + mnorm);
    den_z_eval_device[ tid ] = 0.5*(ps - mnorm);

  }
}


__global__ void eval_uvars_gga_rks_kernel( size_t ntasks, XCDeviceTask* tasks_device) {
  const int batch_idx = blockIdx.z;
  if( batch_idx >= ntasks ) return;
  
  const auto& task = tasks_device[ batch_idx ];
  const auto npts  = task.npts;
  
  const auto*   dden_sx_eval_device = task.dden_sx;
  const auto*   dden_sy_eval_device = task.dden_sy;
  const auto*   dden_sz_eval_device = task.dden_sz;
  auto*         gamma_eval_device   = task.gamma;

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if( tid < npts ) {
    const double dx = dden_sx_eval_device[ tid ];
    const double dy = dden_sy_eval_device[ tid ];
    const double dz = dden_sz_eval_device[ tid ];

    gamma_eval_device[ tid ] = dx*dx + dy*dy + dz*dz;

  }

}

__global__ void eval_uvars_gga_uks_kernel( size_t ntasks, XCDeviceTask* tasks_device) {

  const int batch_idx = blockIdx.z;
  if( batch_idx >= ntasks ) return;

  const auto& task = tasks_device[ batch_idx ];
  const auto npts            = task.npts;

  auto*           den_pos_eval_device   = task.den_s;
  const auto*     den_pos_x_eval_device = task.dden_sx;
  const auto*     den_pos_y_eval_device = task.dden_sy;
  const auto*     den_pos_z_eval_device = task.dden_sz;

  auto*           den_neg_eval_device   = task.den_z;
  const auto*     den_neg_x_eval_device = task.dden_zx;
  const auto*     den_neg_y_eval_device = task.dden_zy;
  const auto*     den_neg_z_eval_device = task.dden_zz;

  auto*     gamma_pp_eval_device  = task.gamma_pp;
  auto*     gamma_pm_eval_device  = task.gamma_pm;
  auto*     gamma_mm_eval_device  = task.gamma_mm;

  const int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if( tid < npts ) {
    const double ps     = den_pos_eval_device[ tid ];
    const double pz     = den_neg_eval_device[ tid ];
    const double dndx   = den_pos_x_eval_device[ tid ];
    const double dndy   = den_pos_y_eval_device[ tid ];
    const double dndz   = den_pos_z_eval_device[ tid ];
    const double dMzdx  = den_neg_x_eval_device[ tid ];
    const double dMzdy  = den_neg_y_eval_device[ tid ];
    const double dMzdz  = den_neg_z_eval_device[ tid ];

    // (del n).(del n)
    const auto dn_sq  = dndx*dndx + dndy*dndy + dndz*dndz;
    // (del Mz).(del Mz)
    const auto dMz_sq = dMzdx*dMzdx + dMzdy*dMzdy + dMzdz*dMzdz;
    // (del n).(del Mz)
    const auto dn_dMz = dndx*dMzdx + dndy*dMzdy + dndz*dMzdz;

    gamma_pp_eval_device[ tid ] = 0.25*(dn_sq + dMz_sq) + 0.5*dn_dMz;
    gamma_pm_eval_device[ tid ] = 0.25*(dn_sq - dMz_sq);
    gamma_mm_eval_device[ tid ] = 0.25*(dn_sq + dMz_sq) - 0.5*dn_dMz;

    den_pos_eval_device[ tid ] = 0.5*(ps + pz);
    den_neg_eval_device[ tid ] = 0.5*(ps - pz);
  }

}

__global__ void eval_uvars_gga_gks_kernel( size_t ntasks, XCDeviceTask* tasks_device) {

  const int batch_idx = blockIdx.z;
  if( batch_idx >= ntasks ) return;

  const auto& task = tasks_device[ batch_idx ];
  const auto npts            = task.npts;

        auto*     den_s_eval_device   = task.den_s;
  const auto*     dden_sx_eval_device = task.dden_sx;
  const auto*     dden_sy_eval_device = task.dden_sy;
  const auto*     dden_sz_eval_device = task.dden_sz;

        auto*     den_z_eval_device   = task.den_z;
  const auto*     dden_zx_eval_device = task.dden_zx;
  const auto*     dden_zy_eval_device = task.dden_zy;
  const auto*     dden_zz_eval_device = task.dden_zz;

  const auto*     den_y_eval_device   = task.den_y;
  const auto*     dden_yx_eval_device = task.dden_yx;
  const auto*     dden_yy_eval_device = task.dden_yy;
  const auto*     dden_yz_eval_device = task.dden_yz;

  const auto*     den_x_eval_device   = task.den_x;
  const auto*     dden_xx_eval_device = task.dden_xx;
  const auto*     dden_xy_eval_device = task.dden_xy;
  const auto*     dden_xz_eval_device = task.dden_xz;

  auto*     gamma_pp_eval_device  = task.gamma_pp;
  auto*     gamma_pm_eval_device  = task.gamma_pm;
  auto*     gamma_mm_eval_device  = task.gamma_mm;

  auto*     H_z_eval_device = task.H_z;
  auto*     H_y_eval_device = task.H_y;
  auto*     H_x_eval_device = task.H_x;
  auto*     K_z_eval_device = task.K_z;
  auto*     K_y_eval_device = task.K_y;
  auto*     K_x_eval_device = task.K_x;

  const double dtolsq = 1e-24;  // TODO: make variable

  const int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if( tid < npts ) {
    const double dndz = dden_sz_eval_device[ tid ];
    const double dndy = dden_sy_eval_device[ tid ];
    const double dndx = dden_sx_eval_device[ tid ];

    const double dMzdz = dden_zz_eval_device[ tid ];
    const double dMzdy = dden_zy_eval_device[ tid ];
    const double dMzdx = dden_zx_eval_device[ tid ];

    const double dMydz = dden_yz_eval_device[ tid ];
    const double dMydy = dden_yy_eval_device[ tid ];
    const double dMydx = dden_yx_eval_device[ tid ];

    const double dMxdz = dden_xz_eval_device[ tid ];
    const double dMxdy = dden_xy_eval_device[ tid ];
    const double dMxdx = dden_xx_eval_device[ tid ];

    const auto ps = den_s_eval_device[ tid ];
    const auto pz = den_z_eval_device[ tid ];
    const auto py = den_y_eval_device[ tid ];
    const auto px = den_x_eval_device[ tid ];

    const auto mtemp = pz*pz + px*px + py*py;
    double mnorm = 0.;

    const auto dels_dot_dels = dndx * dndx + dndy * dndy + dndz * dndz;
    const auto delz_dot_delz = dMzdx * dMzdx + dMzdy * dMzdy + dMzdz * dMzdz;
    const auto delx_dot_delx = dMxdx * dMxdx + dMxdy * dMxdy + dMxdz * dMxdz;
    const auto dely_dot_dely = dMydx * dMydx + dMydy * dMydy + dMydz * dMydz;

    const auto dels_dot_delz = dndx * dMzdx + dndy * dMzdy + dndz * dMzdz;
    const auto dels_dot_delx = dndx * dMxdx + dndy * dMxdy + dndz * dMxdz;
    const auto dels_dot_dely = dndx * dMydx + dndy * dMydy + dndz * dMydz;

    const auto sum = delz_dot_delz + delx_dot_delx + dely_dot_dely;
    const auto s_sum =
               dels_dot_delz * pz + dels_dot_delx * px + dels_dot_dely * py;

    const auto inv_sqsum2 =
        rsqrt(dels_dot_delz * dels_dot_delz + dels_dot_delx * dels_dot_delx +
             dels_dot_dely * dels_dot_dely);
    const auto sqsum2 = 1./inv_sqsum2;

    double sign = 1.;
    if( signbit(s_sum)) 
      sign = -1.;


    if (mtemp > dtolsq) {
      const double inv_mnorm = rsqrt(mtemp);
      mnorm = 1./inv_mnorm;
      K_z_eval_device[ tid ] = pz * inv_mnorm;
      K_y_eval_device[ tid ] = py * inv_mnorm;
      K_x_eval_device[ tid ] = px * inv_mnorm;
      H_z_eval_device[ tid ] = sign * dels_dot_delz * inv_sqsum2;
      H_y_eval_device[ tid ] = sign * dels_dot_dely * inv_sqsum2;
      H_x_eval_device[ tid ] = sign * dels_dot_delx * inv_sqsum2;
    }
    else {
      mnorm = (1. / 3.) * (px + py + pz);
      K_z_eval_device[ tid ] = 1. / 3.;
      K_y_eval_device[ tid ] = 1. / 3.;
      K_x_eval_device[ tid ] = 1. / 3.;

      H_z_eval_device[ tid ] = sign / 3.;
      H_y_eval_device[ tid ] = sign / 3.;
      H_x_eval_device[ tid ] = sign / 3.;
    }

    gamma_pp_eval_device[ tid ] = 0.25*(dels_dot_dels + sum) + 0.5*sign*sqsum2;
    gamma_pm_eval_device[ tid ] = 0.25*(dels_dot_dels - sum);
    gamma_mm_eval_device[ tid ] = 0.25*(dels_dot_dels + sum) - 0.5*sign*sqsum2;

    den_s_eval_device[ tid ] = 0.5*(ps + mnorm);
    den_z_eval_device[ tid ] = 0.5*(ps - mnorm);

  }

}

template <bool need_lapl>
__global__ void eval_uvars_mgga_rks_kernel( size_t           ntasks,
                                       XCDeviceTask* tasks_device ) {

  constexpr auto warp_size = cuda::warp_size;
  //constexpr auto max_warps_per_thread_block = cuda::max_warps_per_thread_block;

  const int batch_idx = blockIdx.z;
  if( batch_idx >= ntasks ) return;

  auto& task = tasks_device[ batch_idx ];

  const auto npts            = task.npts;
  const auto nbf             = task.bfn_screening.nbe;

  auto* tau_eval_device   = task.tau;
  decltype(tau_eval_device) lapl_eval_device = nullptr;
  if constexpr (need_lapl) {
    lapl_eval_device = task.denlapl;
  }

  //const auto* basis_eval_device = task.bf;
  const auto* dbasis_x_eval_device = task.dbfx;
  const auto* dbasis_y_eval_device = task.dbfy;
  const auto* dbasis_z_eval_device = task.dbfz;
  decltype(dbasis_x_eval_device) basis_lapl_eval_device = nullptr;
  if constexpr (need_lapl) {
    basis_lapl_eval_device = task.d2bflapl;
  }

  //const auto* den_basis_prod_device    = task.zmat;
  const auto* den_basis_dx_prod_device = task.xmat_x;
  const auto* den_basis_dy_prod_device = task.xmat_y;
  const auto* den_basis_dz_prod_device = task.xmat_z;
  decltype(den_basis_dx_prod_device) den_basis_prod_device = nullptr;
  if constexpr (need_lapl) {
    den_basis_prod_device = task.zmat;
  }

  __shared__ double den_shared[3+!!need_lapl][warp_size][MGGA_KERNEL_SM_BLOCK+1];

  for ( int bid_x = blockIdx.x * blockDim.x; 
        bid_x < nbf;
        bid_x += blockDim.x * gridDim.x ) {
    
    for ( int bid_y = blockIdx.y * MGGA_KERNEL_SM_BLOCK; 
          bid_y < npts;
          bid_y += MGGA_KERNEL_SM_BLOCK * gridDim.y ) {
        
      for (int sm_y = threadIdx.y; sm_y < MGGA_KERNEL_SM_BLOCK; sm_y += blockDim.y) {
        den_shared[0][threadIdx.x][sm_y] = 0.;
        den_shared[1][threadIdx.x][sm_y] = 0.;
        den_shared[2][threadIdx.x][sm_y] = 0.;
        if constexpr (need_lapl)
          den_shared[3][threadIdx.x][sm_y] = 0.;

        if (bid_y + threadIdx.x < npts and bid_x + sm_y < nbf) { 
          const double* db_x_col = den_basis_dx_prod_device + (bid_x + sm_y)*npts;
          const double* db_y_col = den_basis_dy_prod_device + (bid_x + sm_y)*npts;
          const double* db_z_col = den_basis_dz_prod_device + (bid_x + sm_y)*npts;

          const double* bf_x_col = dbasis_x_eval_device  + (bid_x + sm_y)*npts;
          const double* bf_y_col = dbasis_y_eval_device  + (bid_x + sm_y)*npts;
          const double* bf_z_col = dbasis_z_eval_device  + (bid_x + sm_y)*npts;


          den_shared[0][threadIdx.x][sm_y] = bf_x_col[ bid_y + threadIdx.x ] * db_x_col[ bid_y + threadIdx.x ];
          den_shared[1][threadIdx.x][sm_y] = bf_y_col[ bid_y + threadIdx.x ] * db_y_col[ bid_y + threadIdx.x ];
          den_shared[2][threadIdx.x][sm_y] = bf_z_col[ bid_y + threadIdx.x ] * db_z_col[ bid_y + threadIdx.x ];


          if constexpr (need_lapl) {
            const double* db_col   = den_basis_prod_device  + (bid_x + sm_y)*npts;
            const double* bf_l_col = basis_lapl_eval_device + (bid_x + sm_y)*npts;
            den_shared[3][threadIdx.x][sm_y] = bf_l_col[ bid_y + threadIdx.x ] * db_col[ bid_y + threadIdx.x ];
          }
        }
      }
      __syncthreads();


      for (int sm_y = threadIdx.y; sm_y < MGGA_KERNEL_SM_BLOCK; sm_y += blockDim.y) {
        const int tid_y = bid_y + sm_y;

        register double tx_reg  = den_shared[0][sm_y][threadIdx.x];
        register double ty_reg  = den_shared[1][sm_y][threadIdx.x];
        register double tz_reg  = den_shared[2][sm_y][threadIdx.x];
        // Warp blocks are stored col major
        register double tau_reg = 0.0;
        tau_reg  = 0.5 * cuda::warp_reduce_sum<warp_size>( tx_reg );
        tau_reg += 0.5 * cuda::warp_reduce_sum<warp_size>( ty_reg );
        tau_reg += 0.5 * cuda::warp_reduce_sum<warp_size>( tz_reg );

        register double lapl_reg = 0.0;
        if constexpr (need_lapl) {
          lapl_reg = den_shared[3][sm_y][threadIdx.x];
          lapl_reg = cuda::warp_reduce_sum<warp_size>(lapl_reg);
          lapl_reg = 2. * lapl_reg + 4. * tau_reg;
        }

        if( threadIdx.x == 0 and tid_y < npts ) {
          atomicAdd( tau_eval_device   + tid_y, tau_reg );
          if constexpr (need_lapl) {
            atomicAdd( lapl_eval_device   + tid_y, lapl_reg );
          }
        }
      }
      __syncthreads();
    }
  }
}


#define EVAL_UVARS_KERNEL(xc_approx) \
  hipStream_t stream = queue.queue_as<util::hip_stream>();  \
  dim3 blocks( util::div_ceil( npts_max,  threads.x ),  \
               1, \
               ntasks ); \
  switch ( ks_scheme ) { \
    case RKS: \
      eval_uvars_##xc_approx##_rks_kernel<<< blocks, threads, 0, stream >>>( ntasks, device_tasks ); \
      break; \
    case UKS: \
      eval_uvars_##xc_approx##_uks_kernel<<< blocks, threads, 0, stream >>>( ntasks, device_tasks ); \
      break; \
    case GKS: \
      eval_uvars_##xc_approx##_gks_kernel<<< blocks, threads, 0, stream >>>( ntasks, device_tasks ); \
      break; \
    default: \
      GAUXC_GENERIC_EXCEPTION( "Unexpected KS scheme when attempting to evaluate UV vars" ); \
  } 

void eval_uvars_lda( size_t ntasks, int32_t npts_max, integrator_ks_scheme ks_scheme,
  XCDeviceTask* device_tasks, device_queue queue ) {
  dim3 threads( cuda::max_warps_per_thread_block * cuda::warp_size, 1, 1 );
  EVAL_UVARS_KERNEL(lda);
}



void eval_uvars_gga( size_t ntasks, int32_t npts_max, integrator_ks_scheme ks_scheme,
  XCDeviceTask* device_tasks, device_queue queue ) {
  dim3 threads( GGA_KERNEL_SM_WARPS * cuda::warp_size, 1, 1 );
  EVAL_UVARS_KERNEL(gga);
}



void eval_uvars_mgga( size_t ntasks, size_t npts_total, int32_t nbf_max, 
  int32_t npts_max, bool do_lapl, XCDeviceTask* device_tasks, 
  device_queue queue ) {
  // TODO: This interface should be unified with the lda/gga interfaces
  hipStream_t stream = queue.queue_as<util::hip_stream>();

  // U Variables
  {
  dim3 threads( cuda::warp_size, cuda::max_warps_per_thread_block / 2, 1 );
  dim3 blocks( std::min(uint64_t(4), util::div_ceil( nbf_max, 4 )),
               std::min(uint64_t(MGGA_KERNEL_SM_BLOCK), util::div_ceil( npts_max, MGGA_KERNEL_SM_BLOCK )),
               ntasks );
  if(do_lapl)
    eval_uvars_mgga_rks_kernel<true><<< blocks, threads, 0, stream >>>( ntasks, device_tasks );
  else
    eval_uvars_mgga_rks_kernel<false><<< blocks, threads, 0, stream >>>( ntasks, device_tasks );
  }

  // V variables (GAMMA)
  dim3 threads( cuda::max_threads_per_thread_block );
  dim3 blocks( util::div_ceil( npts_total,  threads.x ),  
               1, 
               ntasks ); 
  eval_uvars_gga_rks_kernel <<< blocks, threads, 0, stream >>>( ntasks, device_tasks );
}







template <density_id den_select>
__global__ void eval_vvar_grad_kern( size_t        ntasks,
                                       XCDeviceTask* tasks_device ) {

  const int batch_idx = blockIdx.z;
  if( batch_idx >= ntasks ) return;

  auto& task = tasks_device[ batch_idx ];

  const auto npts            = task.npts;
  const auto nbf             = task.bfn_screening.nbe;

  double* den_eval_device   = nullptr;
  double* den_x_eval_device = nullptr;
  double* den_y_eval_device = nullptr;
  double* den_z_eval_device = nullptr;

  constexpr auto warp_size = cuda::warp_size;

  if constexpr (den_select == DEN_S) {
    den_eval_device   = task.den_s;
    den_x_eval_device = task.dden_sx;
    den_y_eval_device = task.dden_sy;
    den_z_eval_device = task.dden_sz;
  }
  if constexpr (den_select == DEN_Z) {
    den_eval_device   = task.den_z;
    den_x_eval_device = task.dden_zx;
    den_y_eval_device = task.dden_zy;
    den_z_eval_device = task.dden_zz;
  }
  if constexpr (den_select == DEN_Y) {
    den_eval_device   = task.den_y;
    den_x_eval_device = task.dden_yx;
    den_y_eval_device = task.dden_yy;
    den_z_eval_device = task.dden_yz;
  }
  if constexpr (den_select == DEN_X) {
    den_eval_device   = task.den_x;
    den_x_eval_device = task.dden_xx;
    den_y_eval_device = task.dden_xy;
    den_z_eval_device = task.dden_xz;
  }

  const auto* basis_eval_device = task.bf;
  const auto* dbasis_x_eval_device = task.dbfx;
  const auto* dbasis_y_eval_device = task.dbfy;
  const auto* dbasis_z_eval_device = task.dbfz;

  const auto* den_basis_prod_device = task.zmat;
  
  __shared__ double den_shared[4][warp_size][VVAR_KERNEL_SM_BLOCK+1];

  for ( int bid_x = blockIdx.x * blockDim.x; 
        bid_x < nbf;
        bid_x += blockDim.x * gridDim.x ) {
    
    for ( int bid_y = blockIdx.y * VVAR_KERNEL_SM_BLOCK; 
          bid_y < npts;
          bid_y += VVAR_KERNEL_SM_BLOCK * gridDim.y ) {
        
      for (int sm_y = threadIdx.y; sm_y < VVAR_KERNEL_SM_BLOCK; sm_y += blockDim.y) {
        den_shared[0][threadIdx.x][sm_y] = 0.;
        den_shared[1][threadIdx.x][sm_y] = 0.;
        den_shared[2][threadIdx.x][sm_y] = 0.;
        den_shared[3][threadIdx.x][sm_y] = 0.;

        if (bid_y + threadIdx.x < npts and bid_x + sm_y < nbf) { 
          const double* db_col   = den_basis_prod_device + (bid_x + sm_y)*npts;
          const double* bf_col   = basis_eval_device     + (bid_x + sm_y)*npts;
          const double* bf_x_col = dbasis_x_eval_device  + (bid_x + sm_y)*npts;
          const double* bf_y_col = dbasis_y_eval_device  + (bid_x + sm_y)*npts;
          const double* bf_z_col = dbasis_z_eval_device  + (bid_x + sm_y)*npts;

          den_shared[0][threadIdx.x][sm_y] = bf_col  [ bid_y + threadIdx.x ] * db_col[ bid_y + threadIdx.x ];
          den_shared[1][threadIdx.x][sm_y] = bf_x_col[ bid_y + threadIdx.x ] * db_col[ bid_y + threadIdx.x ];
          den_shared[2][threadIdx.x][sm_y] = bf_y_col[ bid_y + threadIdx.x ] * db_col[ bid_y + threadIdx.x ];
          den_shared[3][threadIdx.x][sm_y] = bf_z_col[ bid_y + threadIdx.x ] * db_col[ bid_y + threadIdx.x ];
        }
      }
      __syncthreads();


      for (int sm_y = threadIdx.y; sm_y < VVAR_KERNEL_SM_BLOCK; sm_y += blockDim.y) {
        const int tid_y = bid_y + sm_y;
        register double den_reg = den_shared[0][sm_y][threadIdx.x];
        register double dx_reg  = den_shared[1][sm_y][threadIdx.x];
        register double dy_reg  = den_shared[2][sm_y][threadIdx.x];
        register double dz_reg  = den_shared[3][sm_y][threadIdx.x];

        // Warp blocks are stored col major
        den_reg =     cuda::warp_reduce_sum<warp_size>( den_reg );
        dx_reg  = 2. * cuda::warp_reduce_sum<warp_size>( dx_reg );
        dy_reg  = 2. * cuda::warp_reduce_sum<warp_size>( dy_reg );
        dz_reg  = 2. * cuda::warp_reduce_sum<warp_size>( dz_reg );


        if( threadIdx.x == 0 and tid_y < npts ) {
          atomicAdd( den_eval_device   + tid_y, den_reg );
          atomicAdd( den_x_eval_device + tid_y, dx_reg  );
          atomicAdd( den_y_eval_device + tid_y, dy_reg  );
          atomicAdd( den_z_eval_device + tid_y, dz_reg  );
        }
      }
      __syncthreads();
    }
  }

}



template <density_id den_select>
__global__ void eval_vvar_kern( size_t        ntasks,
                                       XCDeviceTask* tasks_device ) {

  const int batch_idx = blockIdx.z;
  if( batch_idx >= ntasks ) return;

  auto& task = tasks_device[ batch_idx ];

  const auto npts            = task.npts;
  const auto nbf             = task.bfn_screening.nbe;

  double* den_eval_device   = nullptr;
  // use the "U" variable (+/- for UKS) even though at this point the density (S/Z) is stored
  if constexpr (den_select == DEN_S) den_eval_device = task.den_s;
  if constexpr (den_select == DEN_Z) den_eval_device = task.den_z;
  if constexpr (den_select == DEN_Y) den_eval_device = task.den_y;
  if constexpr (den_select == DEN_X) den_eval_device = task.den_x;

  const auto* basis_eval_device = task.bf;

  const auto* den_basis_prod_device = task.zmat;

  register double den_reg = 0.;

  int start_y = blockIdx.y * blockDim.y + threadIdx.y;

  for (int tid_x = blockIdx.x * blockDim.x + threadIdx.x; 
       tid_x < nbf;
       tid_x += blockDim.x * gridDim.x ) {
    
    for (int tid_y = start_y; 
         tid_y < npts;
         tid_y += blockDim.y * gridDim.y ) {

        const double* bf_col   = basis_eval_device     + tid_x*npts;
        const double* db_col   = den_basis_prod_device + tid_x*npts;

        den_reg += bf_col[ tid_y ]   * db_col[ tid_y ];
    }

  }

  // Warp blocks are stored col major
  constexpr auto warp_size = cuda::warp_size;
  //constexpr auto max_warps_per_thread_block = cuda::max_warps_per_thread_block;
  den_reg = cuda::warp_reduce_sum<warp_size>( den_reg );


  if( threadIdx.x == 0 and start_y < npts ) {
    atomicAdd( den_eval_device   + start_y, den_reg );
  }
  

}




void eval_vvar( size_t ntasks, int32_t nbf_max, int32_t npts_max, bool do_grad, density_id den_select,
  XCDeviceTask* device_tasks, device_queue queue ) {

  hipStream_t stream = queue.queue_as<util::hip_stream>();
  dim3 threads;
  dim3 blocks;
  if( do_grad ) {
    threads = dim3( cuda::warp_size, cuda::max_warps_per_thread_block / 2, 1 );
    blocks = dim3( std::min(uint64_t(4), util::div_ceil( nbf_max, 4 )),
            std::min(uint64_t(16), util::div_ceil( nbf_max, 16 )),
            ntasks );
  } else {
    threads = dim3( cuda::warp_size, cuda::max_warps_per_thread_block, 1 );
    blocks = dim3( util::div_ceil( nbf_max,  threads.x ),
            util::div_ceil( npts_max, threads.y ),
            ntasks );
  }
  switch( den_select ) {
    case DEN_S: 
      if (do_grad)  eval_vvar_grad_kern<DEN_S><<< blocks, threads, 0, stream >>>( ntasks, device_tasks );
      else          eval_vvar_kern<DEN_S><<< blocks, threads, 0, stream >>>( ntasks, device_tasks );
      break;
    case DEN_Z: 
      if (do_grad)  eval_vvar_grad_kern<DEN_Z><<< blocks, threads, 0, stream >>>( ntasks, device_tasks );
      else          eval_vvar_kern<DEN_Z><<< blocks, threads, 0, stream >>>( ntasks, device_tasks );
      break;
    case DEN_Y: 
      if (do_grad)  eval_vvar_grad_kern<DEN_Y><<< blocks, threads, 0, stream >>>( ntasks, device_tasks );
      else          eval_vvar_kern<DEN_Y><<< blocks, threads, 0, stream >>>( ntasks, device_tasks );
      break;
    case DEN_X: 
      if (do_grad)  eval_vvar_grad_kern<DEN_X><<< blocks, threads, 0, stream >>>( ntasks, device_tasks );
      else          eval_vvar_kern<DEN_X><<< blocks, threads, 0, stream >>>( ntasks, device_tasks );
      break;
    default:
      GAUXC_GENERIC_EXCEPTION( "eval_vvar called with improper density selected" );
  }

}





}
