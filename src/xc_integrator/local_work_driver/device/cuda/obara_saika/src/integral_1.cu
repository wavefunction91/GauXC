#include "hip/hip_runtime.h"
/**
 * GauXC Copyright (c) 2020-2024, The Regents of the University of California,
 * through Lawrence Berkeley National Laboratory (subject to receipt of
 * any required approvals from the U.S. Dept. of Energy). All rights reserved.
 *
 * See LICENSE.txt for details
 */
#include <math.h>
#include "../include/gpu/chebyshev_boys_computation.hpp"
#include "config_obara_saika.hpp"
#include "integral_1.hu"

namespace XGPU {
  __inline__ __device__ void dev_integral_1_driver(size_t npts,
				 double *points_x,
				 double *points_y,
				 double *points_z,
                 const int nprim_pairs,
                 const GauXC::PrimitivePair<double>* prim_pairs,
				 double *Xi,
				 int ldX,
				 double *Gi,
				 int ldG,
				 double *weights,
				 double *boys_table) {
    __shared__ double temp[128 * 9];
    
    for(size_t p_outer = blockIdx.x * blockDim.x; p_outer < npts; p_outer += gridDim.x * blockDim.x) {
      double *_point_outer_x = (points_x + p_outer);
      double *_point_outer_y = (points_y + p_outer);
      double *_point_outer_z = (points_z + p_outer);

      size_t p_inner = (threadIdx.x < (npts - p_outer)) ? threadIdx.x : (npts - p_outer);

      for(int i = 0; i < 9; ++i) SCALAR_STORE((temp + i * blockDim.x + threadIdx.x), SCALAR_ZERO());

      for(int ij = 0; ij < nprim_pairs; ++ij) {
	double RHO = prim_pairs[ij].gamma;
	double RHO_INV = prim_pairs[ij].gamma_inv;

	double xA = prim_pairs[ij].P.x;
	double yA = prim_pairs[ij].P.y;
	double zA = prim_pairs[ij].P.z;
	
	constexpr double X_PA = 0.0;
	constexpr double Y_PA = 0.0;
	constexpr double Z_PA = 0.0;

	double eval = prim_pairs[ij].K_coeff_prod;

	// Evaluate T Values
	SCALAR_TYPE xC = SCALAR_LOAD((_point_outer_x + p_inner));
	SCALAR_TYPE yC = SCALAR_LOAD((_point_outer_y + p_inner));
	SCALAR_TYPE zC = SCALAR_LOAD((_point_outer_z + p_inner));

	SCALAR_TYPE X_PC = SCALAR_SUB(xA, xC);
	SCALAR_TYPE Y_PC = SCALAR_SUB(yA, yC);
	SCALAR_TYPE Z_PC = SCALAR_SUB(zA, zC);

	SCALAR_TYPE TVAL = SCALAR_MUL(X_PC, X_PC);
	TVAL = SCALAR_FMA(Y_PC, Y_PC, TVAL);
        TVAL = SCALAR_FMA(Z_PC, Z_PC, TVAL);
	TVAL = SCALAR_MUL(RHO, TVAL);

	SCALAR_TYPE t00, t01, t02, TVAL_inv_e;

	// Evaluate Boys function
	boys_element<2>(&TVAL, &TVAL_inv_e, &t02, boys_table);

	// Evaluate VRR Buffer
	SCALAR_TYPE t10, t11, t20, tx, ty;

	t01 = SCALAR_MUL(SCALAR_ADD(SCALAR_MUL(TVAL, t02), TVAL_inv_e), SCALAR_SET1(0.66666666666666662966));
	t00 = SCALAR_MUL(SCALAR_ADD(SCALAR_MUL(TVAL, t01), TVAL_inv_e), SCALAR_SET1(2.00000000000000000000));
	
	t00 = SCALAR_MUL(eval, t00);
	t01 = SCALAR_MUL(eval, t01);
	t02 = SCALAR_MUL(eval, t02);
	t10 = SCALAR_MUL(X_PA, t00);
	t10 = SCALAR_FNMA(X_PC, t01, t10);
	t11 = SCALAR_MUL(X_PA, t01);
	t11 = SCALAR_FNMA(X_PC, t02, t11);
	tx = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t10);
	SCALAR_STORE((temp + 0 * blockDim.x + threadIdx.x), tx);
	t20 = SCALAR_MUL(X_PA, t10);
	t20 = SCALAR_FNMA(X_PC, t11, t20);
	tx = SCALAR_SUB(t00, t01);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t20 = SCALAR_FMA(tx, ty, t20);
	tx = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t20);
	SCALAR_STORE((temp + 3 * blockDim.x + threadIdx.x), tx);
	t20 = SCALAR_MUL(Y_PA, t10);
	t20 = SCALAR_FNMA(Y_PC, t11, t20);
	tx = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t20);
	SCALAR_STORE((temp + 4 * blockDim.x + threadIdx.x), tx);
	t20 = SCALAR_MUL(Z_PA, t10);
	t20 = SCALAR_FNMA(Z_PC, t11, t20);
	tx = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t20);
	SCALAR_STORE((temp + 5 * blockDim.x + threadIdx.x), tx);
	t10 = SCALAR_MUL(Y_PA, t00);
	t10 = SCALAR_FNMA(Y_PC, t01, t10);
	t11 = SCALAR_MUL(Y_PA, t01);
	t11 = SCALAR_FNMA(Y_PC, t02, t11);
	tx = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t10);
	SCALAR_STORE((temp + 1 * blockDim.x + threadIdx.x), tx);
	t20 = SCALAR_MUL(Y_PA, t10);
	t20 = SCALAR_FNMA(Y_PC, t11, t20);
	tx = SCALAR_SUB(t00, t01);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t20 = SCALAR_FMA(tx, ty, t20);
	tx = SCALAR_LOAD((temp + 6 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t20);
	SCALAR_STORE((temp + 6 * blockDim.x + threadIdx.x), tx);
	t20 = SCALAR_MUL(Z_PA, t10);
	t20 = SCALAR_FNMA(Z_PC, t11, t20);
	tx = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t20);
	SCALAR_STORE((temp + 7 * blockDim.x + threadIdx.x), tx);
	t10 = SCALAR_MUL(Z_PA, t00);
	t10 = SCALAR_FNMA(Z_PC, t01, t10);
	t11 = SCALAR_MUL(Z_PA, t01);
	t11 = SCALAR_FNMA(Z_PC, t02, t11);
	tx = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t10);
	SCALAR_STORE((temp + 2 * blockDim.x + threadIdx.x), tx);
	t20 = SCALAR_MUL(Z_PA, t10);
	t20 = SCALAR_FNMA(Z_PC, t11, t20);
	tx = SCALAR_SUB(t00, t01);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t20 = SCALAR_FMA(tx, ty, t20);
	tx = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t20);
	SCALAR_STORE((temp + 8 * blockDim.x + threadIdx.x), tx);
      }

      if(threadIdx.x < npts - p_outer) {
	double *Xik = (Xi + p_outer + p_inner);
	double *Gik = (Gi + p_outer + p_inner);

	SCALAR_TYPE tx, wg, xik, gik;
	tx  = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
	wg  = SCALAR_LOAD((weights + p_outer + p_inner));

	xik = SCALAR_LOAD((Xik + 0 * ldX));
	gik = SCALAR_LOAD((Gik + 0 * ldG));

	tx = SCALAR_MUL(tx, wg);
	gik = SCALAR_FMA(tx, xik, gik);
	SCALAR_STORE((Gik + 0 * ldG), gik);
	tx  = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
	wg  = SCALAR_LOAD((weights + p_outer + p_inner));

	xik = SCALAR_LOAD((Xik + 0 * ldX));
	gik = SCALAR_LOAD((Gik + 1 * ldG));

	tx = SCALAR_MUL(tx, wg);
	gik = SCALAR_FMA(tx, xik, gik);
	SCALAR_STORE((Gik + 1 * ldG), gik);
	tx  = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
	wg  = SCALAR_LOAD((weights + p_outer + p_inner));

	xik = SCALAR_LOAD((Xik + 0 * ldX));
	gik = SCALAR_LOAD((Gik + 2 * ldG));

	tx = SCALAR_MUL(tx, wg);
	gik = SCALAR_FMA(tx, xik, gik);
	SCALAR_STORE((Gik + 2 * ldG), gik);
	tx  = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
	wg  = SCALAR_LOAD((weights + p_outer + p_inner));

	xik = SCALAR_LOAD((Xik + 1 * ldX));
	gik = SCALAR_LOAD((Gik + 0 * ldG));

	tx = SCALAR_MUL(tx, wg);
	gik = SCALAR_FMA(tx, xik, gik);
	SCALAR_STORE((Gik + 0 * ldG), gik);
	tx  = SCALAR_LOAD((temp + 6 * blockDim.x + threadIdx.x));
	wg  = SCALAR_LOAD((weights + p_outer + p_inner));

	xik = SCALAR_LOAD((Xik + 1 * ldX));
	gik = SCALAR_LOAD((Gik + 1 * ldG));

	tx = SCALAR_MUL(tx, wg);
	gik = SCALAR_FMA(tx, xik, gik);
	SCALAR_STORE((Gik + 1 * ldG), gik);
	tx  = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
	wg  = SCALAR_LOAD((weights + p_outer + p_inner));

	xik = SCALAR_LOAD((Xik + 1 * ldX));
	gik = SCALAR_LOAD((Gik + 2 * ldG));

	tx = SCALAR_MUL(tx, wg);
	gik = SCALAR_FMA(tx, xik, gik);
	SCALAR_STORE((Gik + 2 * ldG), gik);
	tx  = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
	wg  = SCALAR_LOAD((weights + p_outer + p_inner));

	xik = SCALAR_LOAD((Xik + 2 * ldX));
	gik = SCALAR_LOAD((Gik + 0 * ldG));

	tx = SCALAR_MUL(tx, wg);
	gik = SCALAR_FMA(tx, xik, gik);
	SCALAR_STORE((Gik + 0 * ldG), gik);
	tx  = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
	wg  = SCALAR_LOAD((weights + p_outer + p_inner));

	xik = SCALAR_LOAD((Xik + 2 * ldX));
	gik = SCALAR_LOAD((Gik + 1 * ldG));

	tx = SCALAR_MUL(tx, wg);
	gik = SCALAR_FMA(tx, xik, gik);
	SCALAR_STORE((Gik + 1 * ldG), gik);
	tx  = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
	wg  = SCALAR_LOAD((weights + p_outer + p_inner));

	xik = SCALAR_LOAD((Xik + 2 * ldX));
	gik = SCALAR_LOAD((Gik + 2 * ldG));

	tx = SCALAR_MUL(tx, wg);
	gik = SCALAR_FMA(tx, xik, gik);
	SCALAR_STORE((Gik + 2 * ldG), gik);
      }
    }
  }

  __global__ void dev_integral_1(size_t npts,
				   double *points_x,
				   double *points_y,
				   double *points_z,
                   const int nprim_pairs,
                   const GauXC::PrimitivePair<double>* prim_pairs,
				   double *Xi,
				   int ldX,
				   double *Gi,
				   int ldG, 
				   double *weights, 
				   double *boys_table) {
    dev_integral_1_driver( npts, points_x, points_y, points_z, nprim_pairs, prim_pairs, Xi, ldX,
      Gi, ldG, weights, boys_table );
  }

  void integral_1(size_t npts,
		  double *_points_x,	
		  double *_points_y,	
		  double *_points_z,	
          const int nprim_pairs,
          const GauXC::PrimitivePair<double>* prim_pairs,
		  double *Xi,
		  int ldX,
		  double *Gi,
		  int ldG, 
		  double *weights,
		  double *boys_table,
      hipStream_t stream) {
    dev_integral_1<<<320, 128, 0, stream>>>(npts,
				 _points_x,
				 _points_y,
				 _points_z,
         nprim_pairs, prim_pairs,
				 Xi,
				 ldX,
				 Gi,
				 ldG, 
				 weights, 
				 boys_table);
  }

  __global__ void dev_integral_1_batched(
           const GauXC::ShellPairToTaskDevice* sp2task,
           GauXC::XCDeviceTask*                device_tasks,
				   double *boys_table) {

    const int ntask = sp2task->ntask;
    for( int i_task = blockIdx.y; i_task < ntask; i_task += gridDim.y ) {
    
      const auto iT = sp2task->task_idx_device[i_task];
      const auto* task  = device_tasks + iT;
      const auto  npts  = task->npts;

      const auto  i_off = sp2task->task_shell_off_row_device[i_task]*npts;

      dev_integral_1_driver( 
        npts,
        task->points_x,
        task->points_y,
        task->points_z,
        sp2task->nprim_pairs,
        sp2task->prim_pairs_device,
        task->fmat + i_off,
        npts,
        task->gmat + i_off,
        npts,
        task->weights, boys_table );
    }

  }


  void integral_1_batched(size_t ntask_sp,
        const GauXC::ShellPairToTaskDevice* sp2task,
        GauXC::XCDeviceTask*                device_tasks,
		    double *boys_table,
        hipStream_t stream) {

    int nthreads = 128;
    int nblocks_x = 160;
    int nblocks_y = ntask_sp;
    dim3 nblocks(nblocks_x, nblocks_y);
    dev_integral_1_batched<<<nblocks,nthreads,0,stream>>>(
      sp2task, device_tasks, boys_table );

  }
}
