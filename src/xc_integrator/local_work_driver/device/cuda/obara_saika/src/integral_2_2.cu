#include "hip/hip_runtime.h"
/**
 * GauXC Copyright (c) 2020-2024, The Regents of the University of California,
 * through Lawrence Berkeley National Laboratory (subject to receipt of
 * any required approvals from the U.S. Dept. of Energy). All rights reserved.
 *
 * See LICENSE.txt for details
 */
#include <math.h>
#include "../include/gpu/chebyshev_boys_computation.hpp"
#include "config_obara_saika.hpp"
#include "integral_2_2.hu"

#include "task_map_base.hu"

#include "device_specific/cuda_device_constants.hpp"
#include "../../cuda_aos_scheme1.hpp"

namespace XGPU {

using namespace GauXC;

  __inline__ __device__ void dev_integral_2_2_driver(double X_AB,
				   double Y_AB,
				   double Z_AB,
				   size_t npts,
				   double *_points_x,
				   double *_points_y,
				   double *_points_z,
           shell_pair* sp,
				   double *Xi,
				   double *Xj,
				   int ldX,
				   double *Gi,
				   double *Gj,
				   int ldG, 
				   double *weights, 
				   double *boys_table) {
    __shared__ double temp[128 * 31];
    const auto nprim_pairs = sp->nprim_pairs();
    const auto prim_pairs  = sp->prim_pairs();

    __shared__ double outBuffer[128][6];

    for(size_t p_outer = blockIdx.x * blockDim.x; p_outer < npts; p_outer += gridDim.x * blockDim.x) {
      for (int i = 0; i < 6; i++) {
        outBuffer[threadIdx.x][i] = 0.0;
      }

      double *_point_outer_x = (_points_x + p_outer);
      double *_point_outer_y = (_points_y + p_outer);
      double *_point_outer_z = (_points_z + p_outer);

      size_t p_inner = (threadIdx.x < (npts - p_outer)) ? threadIdx.x : (npts - p_outer);

      for(int i = 0; i < 31; ++i) SCALAR_STORE((temp + i * blockDim.x + threadIdx.x), SCALAR_ZERO());

      for(int ij = 0; ij < nprim_pairs; ++ij) {
	double RHO = prim_pairs[ij].gamma;
	double RHO_INV = prim_pairs[ij].gamma_inv;
	double X_PA = prim_pairs[ij].PA.x;
	double Y_PA = prim_pairs[ij].PA.y;
	double Z_PA = prim_pairs[ij].PA.z;

	double xP = prim_pairs[ij].P.x;
	double yP = prim_pairs[ij].P.y;
	double zP = prim_pairs[ij].P.z;

	double eval = prim_pairs[ij].K_coeff_prod;

	// Evaluate T Values
	SCALAR_TYPE xC = SCALAR_LOAD((_point_outer_x + p_inner));
	SCALAR_TYPE yC = SCALAR_LOAD((_point_outer_y + p_inner));
	SCALAR_TYPE zC = SCALAR_LOAD((_point_outer_z + p_inner));

	SCALAR_TYPE X_PC = SCALAR_SUB(xP, xC);
	SCALAR_TYPE Y_PC = SCALAR_SUB(yP, yC);
	SCALAR_TYPE Z_PC = SCALAR_SUB(zP, zC);

	SCALAR_TYPE TVAL = SCALAR_MUL(X_PC, X_PC);
	TVAL = SCALAR_FMA(Y_PC, Y_PC, TVAL);
	TVAL = SCALAR_FMA(Z_PC, Z_PC, TVAL);
	TVAL = SCALAR_MUL(RHO, TVAL);

	SCALAR_TYPE t00, t01, t02, t03, t04, TVAL_inv_e;

	// Evaluate Boys function
	boys_element<4>(&TVAL, &TVAL_inv_e, &t04, boys_table);

	// Evaluate VRR Buffer
	SCALAR_TYPE t10, t11, t12, t13, t20, t21, t22, t30, t31, t40, tx, ty;

	t03 = SCALAR_MUL(SCALAR_ADD(SCALAR_MUL(TVAL, t04), TVAL_inv_e), SCALAR_SET1(0.28571428571428569843));
	t02 = SCALAR_MUL(SCALAR_ADD(SCALAR_MUL(TVAL, t03), TVAL_inv_e), SCALAR_SET1(0.40000000000000002220));
	t01 = SCALAR_MUL(SCALAR_ADD(SCALAR_MUL(TVAL, t02), TVAL_inv_e), SCALAR_SET1(0.66666666666666662966));
	t00 = SCALAR_MUL(SCALAR_ADD(SCALAR_MUL(TVAL, t01), TVAL_inv_e), SCALAR_SET1(2.00000000000000000000));

	t00 = SCALAR_MUL(eval, t00);
	t01 = SCALAR_MUL(eval, t01);
	t02 = SCALAR_MUL(eval, t02);
	t03 = SCALAR_MUL(eval, t03);
	t04 = SCALAR_MUL(eval, t04);
	t10 = SCALAR_MUL(X_PA, t00);
	t10 = SCALAR_FNMA(X_PC, t01, t10);
	t11 = SCALAR_MUL(X_PA, t01);
	t11 = SCALAR_FNMA(X_PC, t02, t11);
	t12 = SCALAR_MUL(X_PA, t02);
	t12 = SCALAR_FNMA(X_PC, t03, t12);
	t13 = SCALAR_MUL(X_PA, t03);
	t13 = SCALAR_FNMA(X_PC, t04, t13);
	t20 = SCALAR_MUL(X_PA, t10);
	t20 = SCALAR_FNMA(X_PC, t11, t20);
	tx = SCALAR_SUB(t00, t01);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t20 = SCALAR_FMA(tx, ty, t20);
	t21 = SCALAR_MUL(X_PA, t11);
	t21 = SCALAR_FNMA(X_PC, t12, t21);
	tx = SCALAR_SUB(t01, t02);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t21 = SCALAR_FMA(tx, ty, t21);
	t22 = SCALAR_MUL(X_PA, t12);
	t22 = SCALAR_FNMA(X_PC, t13, t22);
	tx = SCALAR_SUB(t02, t03);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t22 = SCALAR_FMA(tx, ty, t22);
	tx = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t20);
	SCALAR_STORE((temp + 0 * blockDim.x + threadIdx.x), tx);
	t30 = SCALAR_MUL(X_PA, t20);
	t30 = SCALAR_FNMA(X_PC, t21, t30);
	tx = SCALAR_SUB(t10, t11);
	ty = SCALAR_SET1(0.5 * 2);
	ty = SCALAR_MUL(ty, RHO_INV);
	t30 = SCALAR_FMA(tx, ty, t30);
	t31 = SCALAR_MUL(X_PA, t21);
	t31 = SCALAR_FNMA(X_PC, t22, t31);
	tx = SCALAR_SUB(t11, t12);
	ty = SCALAR_SET1(0.5 * 2);
	ty = SCALAR_MUL(ty, RHO_INV);
	t31 = SCALAR_FMA(tx, ty, t31);
	tx = SCALAR_LOAD((temp + 6 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t30);
	SCALAR_STORE((temp + 6 * blockDim.x + threadIdx.x), tx);
	t40 = SCALAR_MUL(X_PA, t30);
	t40 = SCALAR_FNMA(X_PC, t31, t40);
	tx = SCALAR_SUB(t20, t21);
	ty = SCALAR_SET1(0.5 * 3);
	ty = SCALAR_MUL(ty, RHO_INV);
	t40 = SCALAR_FMA(tx, ty, t40);
	tx = SCALAR_LOAD((temp + 16 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t40);
	SCALAR_STORE((temp + 16 * blockDim.x + threadIdx.x), tx);
	t40 = SCALAR_MUL(Y_PA, t30);
	t40 = SCALAR_FNMA(Y_PC, t31, t40);
	tx = SCALAR_LOAD((temp + 17 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t40);
	SCALAR_STORE((temp + 17 * blockDim.x + threadIdx.x), tx);
	t40 = SCALAR_MUL(Z_PA, t30);
	t40 = SCALAR_FNMA(Z_PC, t31, t40);
	tx = SCALAR_LOAD((temp + 18 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t40);
	SCALAR_STORE((temp + 18 * blockDim.x + threadIdx.x), tx);
	t30 = SCALAR_MUL(Y_PA, t20);
	t30 = SCALAR_FNMA(Y_PC, t21, t30);
	t31 = SCALAR_MUL(Y_PA, t21);
	t31 = SCALAR_FNMA(Y_PC, t22, t31);
	tx = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t30);
	SCALAR_STORE((temp + 7 * blockDim.x + threadIdx.x), tx);
	t40 = SCALAR_MUL(Y_PA, t30);
	t40 = SCALAR_FNMA(Y_PC, t31, t40);
	tx = SCALAR_SUB(t20, t21);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t40 = SCALAR_FMA(tx, ty, t40);
	tx = SCALAR_LOAD((temp + 19 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t40);
	SCALAR_STORE((temp + 19 * blockDim.x + threadIdx.x), tx);
	t40 = SCALAR_MUL(Z_PA, t30);
	t40 = SCALAR_FNMA(Z_PC, t31, t40);
	tx = SCALAR_LOAD((temp + 20 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t40);
	SCALAR_STORE((temp + 20 * blockDim.x + threadIdx.x), tx);
	t30 = SCALAR_MUL(Z_PA, t20);
	t30 = SCALAR_FNMA(Z_PC, t21, t30);
	t31 = SCALAR_MUL(Z_PA, t21);
	t31 = SCALAR_FNMA(Z_PC, t22, t31);
	tx = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t30);
	SCALAR_STORE((temp + 8 * blockDim.x + threadIdx.x), tx);
	t40 = SCALAR_MUL(Z_PA, t30);
	t40 = SCALAR_FNMA(Z_PC, t31, t40);
	tx = SCALAR_SUB(t20, t21);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t40 = SCALAR_FMA(tx, ty, t40);
	tx = SCALAR_LOAD((temp + 21 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t40);
	SCALAR_STORE((temp + 21 * blockDim.x + threadIdx.x), tx);
	t20 = SCALAR_MUL(Y_PA, t10);
	t20 = SCALAR_FNMA(Y_PC, t11, t20);
	t21 = SCALAR_MUL(Y_PA, t11);
	t21 = SCALAR_FNMA(Y_PC, t12, t21);
	t22 = SCALAR_MUL(Y_PA, t12);
	t22 = SCALAR_FNMA(Y_PC, t13, t22);
	tx = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t20);
	SCALAR_STORE((temp + 1 * blockDim.x + threadIdx.x), tx);
	t30 = SCALAR_MUL(Y_PA, t20);
	t30 = SCALAR_FNMA(Y_PC, t21, t30);
	tx = SCALAR_SUB(t10, t11);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t30 = SCALAR_FMA(tx, ty, t30);
	t31 = SCALAR_MUL(Y_PA, t21);
	t31 = SCALAR_FNMA(Y_PC, t22, t31);
	tx = SCALAR_SUB(t11, t12);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t31 = SCALAR_FMA(tx, ty, t31);
	tx = SCALAR_LOAD((temp + 9 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t30);
	SCALAR_STORE((temp + 9 * blockDim.x + threadIdx.x), tx);
	t40 = SCALAR_MUL(Y_PA, t30);
	t40 = SCALAR_FNMA(Y_PC, t31, t40);
	tx = SCALAR_SUB(t20, t21);
	ty = SCALAR_SET1(0.5 * 2);
	ty = SCALAR_MUL(ty, RHO_INV);
	t40 = SCALAR_FMA(tx, ty, t40);
	tx = SCALAR_LOAD((temp + 22 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t40);
	SCALAR_STORE((temp + 22 * blockDim.x + threadIdx.x), tx);
	t40 = SCALAR_MUL(Z_PA, t30);
	t40 = SCALAR_FNMA(Z_PC, t31, t40);
	tx = SCALAR_LOAD((temp + 23 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t40);
	SCALAR_STORE((temp + 23 * blockDim.x + threadIdx.x), tx);
	t30 = SCALAR_MUL(Z_PA, t20);
	t30 = SCALAR_FNMA(Z_PC, t21, t30);
	t31 = SCALAR_MUL(Z_PA, t21);
	t31 = SCALAR_FNMA(Z_PC, t22, t31);
	tx = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t30);
	SCALAR_STORE((temp + 10 * blockDim.x + threadIdx.x), tx);
	t40 = SCALAR_MUL(Z_PA, t30);
	t40 = SCALAR_FNMA(Z_PC, t31, t40);
	tx = SCALAR_SUB(t20, t21);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t40 = SCALAR_FMA(tx, ty, t40);
	tx = SCALAR_LOAD((temp + 24 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t40);
	SCALAR_STORE((temp + 24 * blockDim.x + threadIdx.x), tx);
	t20 = SCALAR_MUL(Z_PA, t10);
	t20 = SCALAR_FNMA(Z_PC, t11, t20);
	t21 = SCALAR_MUL(Z_PA, t11);
	t21 = SCALAR_FNMA(Z_PC, t12, t21);
	t22 = SCALAR_MUL(Z_PA, t12);
	t22 = SCALAR_FNMA(Z_PC, t13, t22);
	tx = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t20);
	SCALAR_STORE((temp + 2 * blockDim.x + threadIdx.x), tx);
	t30 = SCALAR_MUL(Z_PA, t20);
	t30 = SCALAR_FNMA(Z_PC, t21, t30);
	tx = SCALAR_SUB(t10, t11);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t30 = SCALAR_FMA(tx, ty, t30);
	t31 = SCALAR_MUL(Z_PA, t21);
	t31 = SCALAR_FNMA(Z_PC, t22, t31);
	tx = SCALAR_SUB(t11, t12);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t31 = SCALAR_FMA(tx, ty, t31);
	tx = SCALAR_LOAD((temp + 11 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t30);
	SCALAR_STORE((temp + 11 * blockDim.x + threadIdx.x), tx);
	t40 = SCALAR_MUL(Z_PA, t30);
	t40 = SCALAR_FNMA(Z_PC, t31, t40);
	tx = SCALAR_SUB(t20, t21);
	ty = SCALAR_SET1(0.5 * 2);
	ty = SCALAR_MUL(ty, RHO_INV);
	t40 = SCALAR_FMA(tx, ty, t40);
	tx = SCALAR_LOAD((temp + 25 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t40);
	SCALAR_STORE((temp + 25 * blockDim.x + threadIdx.x), tx);
	t10 = SCALAR_MUL(Y_PA, t00);
	t10 = SCALAR_FNMA(Y_PC, t01, t10);
	t11 = SCALAR_MUL(Y_PA, t01);
	t11 = SCALAR_FNMA(Y_PC, t02, t11);
	t12 = SCALAR_MUL(Y_PA, t02);
	t12 = SCALAR_FNMA(Y_PC, t03, t12);
	t13 = SCALAR_MUL(Y_PA, t03);
	t13 = SCALAR_FNMA(Y_PC, t04, t13);
	t20 = SCALAR_MUL(Y_PA, t10);
	t20 = SCALAR_FNMA(Y_PC, t11, t20);
	tx = SCALAR_SUB(t00, t01);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t20 = SCALAR_FMA(tx, ty, t20);
	t21 = SCALAR_MUL(Y_PA, t11);
	t21 = SCALAR_FNMA(Y_PC, t12, t21);
	tx = SCALAR_SUB(t01, t02);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t21 = SCALAR_FMA(tx, ty, t21);
	t22 = SCALAR_MUL(Y_PA, t12);
	t22 = SCALAR_FNMA(Y_PC, t13, t22);
	tx = SCALAR_SUB(t02, t03);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t22 = SCALAR_FMA(tx, ty, t22);
	tx = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t20);
	SCALAR_STORE((temp + 3 * blockDim.x + threadIdx.x), tx);
	t30 = SCALAR_MUL(Y_PA, t20);
	t30 = SCALAR_FNMA(Y_PC, t21, t30);
	tx = SCALAR_SUB(t10, t11);
	ty = SCALAR_SET1(0.5 * 2);
	ty = SCALAR_MUL(ty, RHO_INV);
	t30 = SCALAR_FMA(tx, ty, t30);
	t31 = SCALAR_MUL(Y_PA, t21);
	t31 = SCALAR_FNMA(Y_PC, t22, t31);
	tx = SCALAR_SUB(t11, t12);
	ty = SCALAR_SET1(0.5 * 2);
	ty = SCALAR_MUL(ty, RHO_INV);
	t31 = SCALAR_FMA(tx, ty, t31);
	tx = SCALAR_LOAD((temp + 12 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t30);
	SCALAR_STORE((temp + 12 * blockDim.x + threadIdx.x), tx);
	t40 = SCALAR_MUL(Y_PA, t30);
	t40 = SCALAR_FNMA(Y_PC, t31, t40);
	tx = SCALAR_SUB(t20, t21);
	ty = SCALAR_SET1(0.5 * 3);
	ty = SCALAR_MUL(ty, RHO_INV);
	t40 = SCALAR_FMA(tx, ty, t40);
	tx = SCALAR_LOAD((temp + 26 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t40);
	SCALAR_STORE((temp + 26 * blockDim.x + threadIdx.x), tx);
	t40 = SCALAR_MUL(Z_PA, t30);
	t40 = SCALAR_FNMA(Z_PC, t31, t40);
	tx = SCALAR_LOAD((temp + 27 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t40);
	SCALAR_STORE((temp + 27 * blockDim.x + threadIdx.x), tx);
	t30 = SCALAR_MUL(Z_PA, t20);
	t30 = SCALAR_FNMA(Z_PC, t21, t30);
	t31 = SCALAR_MUL(Z_PA, t21);
	t31 = SCALAR_FNMA(Z_PC, t22, t31);
	tx = SCALAR_LOAD((temp + 13 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t30);
	SCALAR_STORE((temp + 13 * blockDim.x + threadIdx.x), tx);
	t40 = SCALAR_MUL(Z_PA, t30);
	t40 = SCALAR_FNMA(Z_PC, t31, t40);
	tx = SCALAR_SUB(t20, t21);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t40 = SCALAR_FMA(tx, ty, t40);
	tx = SCALAR_LOAD((temp + 28 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t40);
	SCALAR_STORE((temp + 28 * blockDim.x + threadIdx.x), tx);
	t20 = SCALAR_MUL(Z_PA, t10);
	t20 = SCALAR_FNMA(Z_PC, t11, t20);
	t21 = SCALAR_MUL(Z_PA, t11);
	t21 = SCALAR_FNMA(Z_PC, t12, t21);
	t22 = SCALAR_MUL(Z_PA, t12);
	t22 = SCALAR_FNMA(Z_PC, t13, t22);
	tx = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t20);
	SCALAR_STORE((temp + 4 * blockDim.x + threadIdx.x), tx);
	t30 = SCALAR_MUL(Z_PA, t20);
	t30 = SCALAR_FNMA(Z_PC, t21, t30);
	tx = SCALAR_SUB(t10, t11);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t30 = SCALAR_FMA(tx, ty, t30);
	t31 = SCALAR_MUL(Z_PA, t21);
	t31 = SCALAR_FNMA(Z_PC, t22, t31);
	tx = SCALAR_SUB(t11, t12);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t31 = SCALAR_FMA(tx, ty, t31);
	tx = SCALAR_LOAD((temp + 14 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t30);
	SCALAR_STORE((temp + 14 * blockDim.x + threadIdx.x), tx);
	t40 = SCALAR_MUL(Z_PA, t30);
	t40 = SCALAR_FNMA(Z_PC, t31, t40);
	tx = SCALAR_SUB(t20, t21);
	ty = SCALAR_SET1(0.5 * 2);
	ty = SCALAR_MUL(ty, RHO_INV);
	t40 = SCALAR_FMA(tx, ty, t40);
	tx = SCALAR_LOAD((temp + 29 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t40);
	SCALAR_STORE((temp + 29 * blockDim.x + threadIdx.x), tx);
	t10 = SCALAR_MUL(Z_PA, t00);
	t10 = SCALAR_FNMA(Z_PC, t01, t10);
	t11 = SCALAR_MUL(Z_PA, t01);
	t11 = SCALAR_FNMA(Z_PC, t02, t11);
	t12 = SCALAR_MUL(Z_PA, t02);
	t12 = SCALAR_FNMA(Z_PC, t03, t12);
	t13 = SCALAR_MUL(Z_PA, t03);
	t13 = SCALAR_FNMA(Z_PC, t04, t13);
	t20 = SCALAR_MUL(Z_PA, t10);
	t20 = SCALAR_FNMA(Z_PC, t11, t20);
	tx = SCALAR_SUB(t00, t01);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t20 = SCALAR_FMA(tx, ty, t20);
	t21 = SCALAR_MUL(Z_PA, t11);
	t21 = SCALAR_FNMA(Z_PC, t12, t21);
	tx = SCALAR_SUB(t01, t02);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t21 = SCALAR_FMA(tx, ty, t21);
	t22 = SCALAR_MUL(Z_PA, t12);
	t22 = SCALAR_FNMA(Z_PC, t13, t22);
	tx = SCALAR_SUB(t02, t03);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t22 = SCALAR_FMA(tx, ty, t22);
	tx = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t20);
	SCALAR_STORE((temp + 5 * blockDim.x + threadIdx.x), tx);
	t30 = SCALAR_MUL(Z_PA, t20);
	t30 = SCALAR_FNMA(Z_PC, t21, t30);
	tx = SCALAR_SUB(t10, t11);
	ty = SCALAR_SET1(0.5 * 2);
	ty = SCALAR_MUL(ty, RHO_INV);
	t30 = SCALAR_FMA(tx, ty, t30);
	t31 = SCALAR_MUL(Z_PA, t21);
	t31 = SCALAR_FNMA(Z_PC, t22, t31);
	tx = SCALAR_SUB(t11, t12);
	ty = SCALAR_SET1(0.5 * 2);
	ty = SCALAR_MUL(ty, RHO_INV);
	t31 = SCALAR_FMA(tx, ty, t31);
	tx = SCALAR_LOAD((temp + 15 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t30);
	SCALAR_STORE((temp + 15 * blockDim.x + threadIdx.x), tx);
	t40 = SCALAR_MUL(Z_PA, t30);
	t40 = SCALAR_FNMA(Z_PC, t31, t40);
	tx = SCALAR_SUB(t20, t21);
	ty = SCALAR_SET1(0.5 * 3);
	ty = SCALAR_MUL(ty, RHO_INV);
	t40 = SCALAR_FMA(tx, ty, t40);
	tx = SCALAR_LOAD((temp + 30 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t40);
	SCALAR_STORE((temp + 30 * blockDim.x + threadIdx.x), tx);
      }

      if(threadIdx.x < npts - p_outer) {
	double *Xik = (Xi + p_outer + p_inner);
	double *Xjk = (Xj + p_outer + p_inner);
	double *Gik = (Gi + p_outer + p_inner);
	double *Gjk = (Gj + p_outer + p_inner);

	SCALAR_TYPE const_value_v = SCALAR_LOAD((weights + p_outer + p_inner));

	double const_value, X_ABp, Y_ABp, Z_ABp, comb_m_i, comb_n_j, comb_p_k;
	SCALAR_TYPE const_value_w;
	SCALAR_TYPE tx, ty, tz, tw, t0, t1, t2, t3, t4, t5;

  #if 0
	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t0 = SCALAR_LOAD((temp + 16 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t1 = SCALAR_LOAD((temp + 17 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t2 = SCALAR_LOAD((temp + 18 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t3 = SCALAR_LOAD((temp + 19 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t4 = SCALAR_LOAD((temp + 20 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t5 = SCALAR_LOAD((temp + 21 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 0 * ldG), tw);
	X_ABp = SCALAR_MUL(X_ABp, X_AB); comb_m_i = SCALAR_MUL(comb_m_i * 2, SCALAR_RECIPROCAL(1));
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t0 = SCALAR_LOAD((temp + 6 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t1 = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t2 = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t3 = SCALAR_LOAD((temp + 9 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t4 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t5 = SCALAR_LOAD((temp + 11 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 0 * ldG), tw);
	X_ABp = SCALAR_MUL(X_ABp, X_AB); comb_m_i = SCALAR_MUL(comb_m_i * 1, SCALAR_RECIPROCAL(2));
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t1 = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t2 = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t3 = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t4 = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t5 = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 0 * ldG), tw);
	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t0 = SCALAR_LOAD((temp + 17 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t1 = SCALAR_LOAD((temp + 19 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t2 = SCALAR_LOAD((temp + 20 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t3 = SCALAR_LOAD((temp + 22 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t4 = SCALAR_LOAD((temp + 23 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t5 = SCALAR_LOAD((temp + 24 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	Y_ABp = SCALAR_MUL(Y_ABp, Y_AB); comb_n_j = SCALAR_MUL(comb_n_j * 1, SCALAR_RECIPROCAL(1));
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t0 = SCALAR_LOAD((temp + 6 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t1 = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t2 = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t3 = SCALAR_LOAD((temp + 9 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t4 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t5 = SCALAR_LOAD((temp + 11 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	X_ABp = SCALAR_MUL(X_ABp, X_AB); comb_m_i = SCALAR_MUL(comb_m_i * 1, SCALAR_RECIPROCAL(1));
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t0 = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t1 = SCALAR_LOAD((temp + 9 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t2 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t3 = SCALAR_LOAD((temp + 12 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t4 = SCALAR_LOAD((temp + 13 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t5 = SCALAR_LOAD((temp + 14 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	Y_ABp = SCALAR_MUL(Y_ABp, Y_AB); comb_n_j = SCALAR_MUL(comb_n_j * 1, SCALAR_RECIPROCAL(1));
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t1 = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t2 = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t3 = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t4 = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t5 = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);
	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t0 = SCALAR_LOAD((temp + 18 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t1 = SCALAR_LOAD((temp + 20 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t2 = SCALAR_LOAD((temp + 21 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t3 = SCALAR_LOAD((temp + 23 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t4 = SCALAR_LOAD((temp + 24 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t5 = SCALAR_LOAD((temp + 25 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 1, SCALAR_RECIPROCAL(1));
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t0 = SCALAR_LOAD((temp + 6 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t1 = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t2 = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t3 = SCALAR_LOAD((temp + 9 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t4 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t5 = SCALAR_LOAD((temp + 11 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	X_ABp = SCALAR_MUL(X_ABp, X_AB); comb_m_i = SCALAR_MUL(comb_m_i * 1, SCALAR_RECIPROCAL(1));
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t0 = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t1 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t2 = SCALAR_LOAD((temp + 11 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t3 = SCALAR_LOAD((temp + 13 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t4 = SCALAR_LOAD((temp + 14 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t5 = SCALAR_LOAD((temp + 15 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 1, SCALAR_RECIPROCAL(1));
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t1 = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t2 = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t3 = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t4 = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t5 = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 3 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 3 * ldG));
	t0 = SCALAR_LOAD((temp + 19 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 3 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 3 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 3 * ldG));
	t1 = SCALAR_LOAD((temp + 22 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 3 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 3 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 3 * ldG));
	t2 = SCALAR_LOAD((temp + 23 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 3 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 3 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 3 * ldG));
	t3 = SCALAR_LOAD((temp + 26 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 3 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 3 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 3 * ldG));
	t4 = SCALAR_LOAD((temp + 27 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 3 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 3 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 3 * ldG));
	t5 = SCALAR_LOAD((temp + 28 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 3 * ldG), tw);
	Y_ABp = SCALAR_MUL(Y_ABp, Y_AB); comb_n_j = SCALAR_MUL(comb_n_j * 2, SCALAR_RECIPROCAL(1));
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 3 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 3 * ldG));
	t0 = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 3 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 3 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 3 * ldG));
	t1 = SCALAR_LOAD((temp + 9 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 3 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 3 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 3 * ldG));
	t2 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 3 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 3 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 3 * ldG));
	t3 = SCALAR_LOAD((temp + 12 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 3 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 3 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 3 * ldG));
	t4 = SCALAR_LOAD((temp + 13 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 3 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 3 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 3 * ldG));
	t5 = SCALAR_LOAD((temp + 14 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 3 * ldG), tw);
	Y_ABp = SCALAR_MUL(Y_ABp, Y_AB); comb_n_j = SCALAR_MUL(comb_n_j * 1, SCALAR_RECIPROCAL(2));
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 3 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 3 * ldG));
	t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 3 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 3 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 3 * ldG));
	t1 = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 3 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 3 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 3 * ldG));
	t2 = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 3 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 3 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 3 * ldG));
	t3 = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 3 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 3 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 3 * ldG));
	t4 = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 3 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 3 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 3 * ldG));
	t5 = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 3 * ldG), tw);
	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t0 = SCALAR_LOAD((temp + 20 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t1 = SCALAR_LOAD((temp + 23 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t2 = SCALAR_LOAD((temp + 24 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t3 = SCALAR_LOAD((temp + 27 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t4 = SCALAR_LOAD((temp + 28 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t5 = SCALAR_LOAD((temp + 29 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 1, SCALAR_RECIPROCAL(1));
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t0 = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t1 = SCALAR_LOAD((temp + 9 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t2 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t3 = SCALAR_LOAD((temp + 12 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t4 = SCALAR_LOAD((temp + 13 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t5 = SCALAR_LOAD((temp + 14 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	Y_ABp = SCALAR_MUL(Y_ABp, Y_AB); comb_n_j = SCALAR_MUL(comb_n_j * 1, SCALAR_RECIPROCAL(1));
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t0 = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t1 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t2 = SCALAR_LOAD((temp + 11 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t3 = SCALAR_LOAD((temp + 13 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t4 = SCALAR_LOAD((temp + 14 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t5 = SCALAR_LOAD((temp + 15 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 1, SCALAR_RECIPROCAL(1));
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t1 = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t2 = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t3 = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t4 = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 4 * ldG));
	t5 = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 4 * ldG), tw);
	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 5 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 5 * ldG));
	t0 = SCALAR_LOAD((temp + 21 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 5 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 5 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 5 * ldG));
	t1 = SCALAR_LOAD((temp + 24 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 5 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 5 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 5 * ldG));
	t2 = SCALAR_LOAD((temp + 25 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 5 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 5 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 5 * ldG));
	t3 = SCALAR_LOAD((temp + 28 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 5 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 5 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 5 * ldG));
	t4 = SCALAR_LOAD((temp + 29 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 5 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 5 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 5 * ldG));
	t5 = SCALAR_LOAD((temp + 30 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 5 * ldG), tw);
	Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 2, SCALAR_RECIPROCAL(1));
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 5 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 5 * ldG));
	t0 = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 5 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 5 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 5 * ldG));
	t1 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 5 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 5 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 5 * ldG));
	t2 = SCALAR_LOAD((temp + 11 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 5 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 5 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 5 * ldG));
	t3 = SCALAR_LOAD((temp + 13 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 5 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 5 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 5 * ldG));
	t4 = SCALAR_LOAD((temp + 14 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 5 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 5 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 5 * ldG));
	t5 = SCALAR_LOAD((temp + 15 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 5 * ldG), tw);
	Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 1, SCALAR_RECIPROCAL(2));
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 5 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 5 * ldG));
	t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);
	atomicAdd((Gjk + 5 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 5 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 5 * ldG));
	t1 = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);
	atomicAdd((Gjk + 5 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 5 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 5 * ldG));
	t2 = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 5 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 5 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 5 * ldG));
	t3 = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	atomicAdd((Gik + 3 * ldG), tz);
	atomicAdd((Gjk + 5 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 5 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 5 * ldG));
	t4 = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	atomicAdd((Gik + 4 * ldG), tz);
	atomicAdd((Gjk + 5 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 5 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 5 * ldG));
	t5 = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	atomicAdd((Gik + 5 * ldG), tz);
	atomicAdd((Gjk + 5 * ldG), tw);
  #else
	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	t0 = SCALAR_LOAD((temp + 16 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_MUL(tx, t0);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 17 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 18 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 19 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 20 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 21 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
                                
	X_ABp = SCALAR_MUL(X_ABp, X_AB); comb_m_i = SCALAR_MUL(comb_m_i * 2, SCALAR_RECIPROCAL(1));
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	t0 = SCALAR_LOAD((temp + 6 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_FMA(tx, t0, tw);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 9 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 11 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
                                
	X_ABp = SCALAR_MUL(X_ABp, X_AB); comb_m_i = SCALAR_MUL(comb_m_i * 1, SCALAR_RECIPROCAL(2));
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_FMA(tx, t0, tw);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
	atomicAdd((Gjk + 0 * ldG), tw);
  


	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	t0 = SCALAR_LOAD((temp + 17 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_MUL(tx, t0);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 19 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 20 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 22 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 23 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 24 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
                                
	Y_ABp = SCALAR_MUL(Y_ABp, Y_AB); comb_n_j = SCALAR_MUL(comb_n_j * 1, SCALAR_RECIPROCAL(1));
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	t0 = SCALAR_LOAD((temp + 6 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_FMA(tx, t0, tw);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 9 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 11 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
                                
	X_ABp = SCALAR_MUL(X_ABp, X_AB); comb_m_i = SCALAR_MUL(comb_m_i * 1, SCALAR_RECIPROCAL(1));
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	t0 = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_FMA(tx, t0, tw);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 9 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 12 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 13 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 14 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
                                
	Y_ABp = SCALAR_MUL(Y_ABp, Y_AB); comb_n_j = SCALAR_MUL(comb_n_j * 1, SCALAR_RECIPROCAL(1));
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_FMA(tx, t0, tw);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
	atomicAdd((Gjk + 1 * ldG), tw);




	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	t0 = SCALAR_LOAD((temp + 18 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_MUL(tx, t0);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 20 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 21 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 23 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 24 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 25 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
                                
	Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 1, SCALAR_RECIPROCAL(1));
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	t0 = SCALAR_LOAD((temp + 6 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_FMA(tx, t0, tw);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 9 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 11 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
                                
	X_ABp = SCALAR_MUL(X_ABp, X_AB); comb_m_i = SCALAR_MUL(comb_m_i * 1, SCALAR_RECIPROCAL(1));
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	t0 = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_FMA(tx, t0, tw);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 11 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 13 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 14 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 15 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
                                
	Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 1, SCALAR_RECIPROCAL(1));
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_FMA(tx, t0, tw);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
	atomicAdd((Gjk + 2 * ldG), tw);





	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 3 * ldX));
	t0 = SCALAR_LOAD((temp + 19 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_MUL(tx, t0);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 22 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 23 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 26 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 27 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 28 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
                                
	Y_ABp = SCALAR_MUL(Y_ABp, Y_AB); comb_n_j = SCALAR_MUL(comb_n_j * 2, SCALAR_RECIPROCAL(1));
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	t0 = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_FMA(tx, t0, tw);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 9 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 12 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 13 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 14 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
                                
	Y_ABp = SCALAR_MUL(Y_ABp, Y_AB); comb_n_j = SCALAR_MUL(comb_n_j * 1, SCALAR_RECIPROCAL(2));
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_FMA(tx, t0, tw);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
	atomicAdd((Gjk + 3 * ldG), tw);



	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 4 * ldX));
	t0 = SCALAR_LOAD((temp + 20 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_MUL(tx, t0);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 23 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 24 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 27 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 28 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 29 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
                                
	Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 1, SCALAR_RECIPROCAL(1));
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	t0 = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_FMA(tx, t0, tw);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 9 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 12 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 13 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 14 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
                                
	Y_ABp = SCALAR_MUL(Y_ABp, Y_AB); comb_n_j = SCALAR_MUL(comb_n_j * 1, SCALAR_RECIPROCAL(1));
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	t0 = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_FMA(tx, t0, tw);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 11 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 13 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 14 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 15 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
                                
	Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 1, SCALAR_RECIPROCAL(1));
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_FMA(tx, t0, tw);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
	atomicAdd((Gjk + 4 * ldG), tw);




	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 5 * ldX));
	t0 = SCALAR_LOAD((temp + 21 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_MUL(tx, t0);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 24 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 25 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 28 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 29 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 30 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
                                
	Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 2, SCALAR_RECIPROCAL(1));
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	t0 = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_FMA(tx, t0, tw);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 11 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 13 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 14 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 15 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
                                
	Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 1, SCALAR_RECIPROCAL(2));
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_FMA(tx, t0, tw);
	//atomicAdd((Gik + 0 * ldG), tz);
	outBuffer[threadIdx.x][0] += tz;
                                
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	//atomicAdd((Gik + 1 * ldG), tz);
	outBuffer[threadIdx.x][1] += tz;
                                
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	//atomicAdd((Gik + 2 * ldG), tz);
	outBuffer[threadIdx.x][2] += tz;
                                
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	t3 = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_MUL(ty, t3);
	tw = SCALAR_FMA(tx, t3, tw);
	//atomicAdd((Gik + 3 * ldG), tz);
	outBuffer[threadIdx.x][3] += tz;
                                
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	t4 = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_MUL(ty, t4);
	tw = SCALAR_FMA(tx, t4, tw);
	//atomicAdd((Gik + 4 * ldG), tz);
	outBuffer[threadIdx.x][4] += tz;
                                
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	t5 = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_MUL(ty, t5);
	tw = SCALAR_FMA(tx, t5, tw);
	//atomicAdd((Gik + 5 * ldG), tz);
	outBuffer[threadIdx.x][5] += tz;
	atomicAdd((Gjk + 5 * ldG), tw);

	atomicAdd((Gik + 0 * ldG), outBuffer[threadIdx.x][0]);
	atomicAdd((Gik + 1 * ldG), outBuffer[threadIdx.x][1]);
	atomicAdd((Gik + 2 * ldG), outBuffer[threadIdx.x][2]);
	atomicAdd((Gik + 3 * ldG), outBuffer[threadIdx.x][3]);
	atomicAdd((Gik + 4 * ldG), outBuffer[threadIdx.x][4]);
	atomicAdd((Gik + 5 * ldG), outBuffer[threadIdx.x][5]);

  #endif
      }
    }
  }

  __global__ void dev_integral_2_2(
           double X_AB,
				   double Y_AB,
				   double Z_AB,
           size_t npts,
				   double *points_x,
				   double *points_y,
				   double *points_z,
           shell_pair* sp,
				   double *Xi,
				   double *Xj,
				   int ldX,
				   double *Gi,
				   double *Gj,
				   int ldG, 
				   double *weights, 
				   double *boys_table) {
    dev_integral_2_2_driver( X_AB, Y_AB, Z_AB, npts, points_x, points_y, 
      points_z, sp, Xi, Xj, ldX, Gi, Gj, ldG, weights, boys_table );
  }

  void integral_2_2(double X_AB,
		    double Y_AB,
		    double Z_AB,
		    size_t npts,
		    double *points_x,
		    double *points_y,
		    double *points_z,
        shell_pair* sp,
		    double *Xi,
		    double *Xj,
		    int ldX,
		    double *Gi,
		    double *Gj,
		    int ldG, 
		    double *weights, 
		  double *boys_table,
      hipStream_t stream) {
    dev_integral_2_2<<<320, 128, 0, stream>>>(X_AB,
				   Y_AB,
				   Z_AB,
				   npts,
				   points_x,
				   points_y,
				   points_z,
           sp,
				   Xi,
				   Xj,
				   ldX,
				   Gi,
				   Gj,
				   ldG, 
				   weights,
				   boys_table);
  }

  __inline__ __device__ void dev_integral_2_2_batched_driver (
           double X_AB,
				   double Y_AB,
				   double Z_AB,
           const GauXC::ShellPairToTaskDevice* sp2task,
           GauXC::XCDeviceTask*                device_tasks,
				   double *boys_table) {

    //if (sp2task->shell_pair_device->nprim_pairs() == 0) return;
    const int ntask = sp2task->ntask;
    for( int i_task = blockIdx.y; i_task < ntask; i_task += gridDim.y ) {
    
      const auto iT = sp2task->task_idx_device[i_task];
      const auto* task  = device_tasks + iT;
      const auto  npts  = task->npts;

      const auto  i_off = sp2task->task_shell_off_row_device[i_task]*npts;
      const auto  j_off = sp2task->task_shell_off_col_device[i_task]*npts;


      dev_integral_2_2_driver( 
        X_AB, Y_AB, Z_AB,
        npts,
        task->points_x,
        task->points_y,
        task->points_z,
        sp2task->shell_pair_device,
        task->fmat + i_off,
        task->fmat + j_off,
        npts,
        task->gmat + i_off,
        task->gmat + j_off,
        npts,
        task->weights, boys_table );
    }

  }

  __global__ void dev_integral_2_2_batched(
           double X_AB,
				   double Y_AB,
				   double Z_AB,
           const GauXC::ShellPairToTaskDevice* sp2task,
           GauXC::XCDeviceTask*                device_tasks,
				   double *boys_table) {
   dev_integral_2_2_batched_driver(X_AB,Y_AB,Z_AB,sp2task,device_tasks,boys_table);
 }



  void integral_2_2_batched(size_t ntask_sp,
        double X_AB,
				double Y_AB,
				double Z_AB,
        const GauXC::ShellPairToTaskDevice* sp2task,
        GauXC::XCDeviceTask*                device_tasks,
		    double *boys_table,
        hipStream_t stream) {

    int nthreads = 128;
    int nblocks_x = 160;
    int nblocks_y = ntask_sp;
    dim3 nblocks(nblocks_x, nblocks_y);

    dev_integral_2_2_batched<<<nblocks,nthreads,0,stream>>>(
      X_AB, Y_AB, Z_AB, sp2task, device_tasks, boys_table );

  }



  __global__ void dev_integral_2_2_shell_batched(
           int nsp,
           const GauXC::ShellPairToTaskDevice* sp2task,
           GauXC::XCDeviceTask*                device_tasks,
				   double *boys_table) {

   for( int i = blockIdx.z; i < nsp; i += gridDim.z ) {
     auto sp = sp2task + i;
     const auto X_AB = sp->X_AB;
     const auto Y_AB = sp->Y_AB;
     const auto Z_AB = sp->Z_AB;
     dev_integral_2_2_batched_driver(X_AB,Y_AB,Z_AB,sp,device_tasks,boys_table);
   }
 }

  void integral_2_2_shell_batched(
        size_t nsp,
        size_t max_ntask,
        const GauXC::ShellPairToTaskDevice* sp2task,
        GauXC::XCDeviceTask*                device_tasks,
		    double *boys_table,
        hipStream_t stream) {

    int nthreads = 128;
    int nblocks_x = 1;
    int nblocks_y = max_ntask;
    int nblocks_z = nsp;
    dim3 nblocks(nblocks_x, nblocks_y, nblocks_z);
    dev_integral_2_2_shell_batched<<<nblocks,nthreads,0,stream>>>(
      nsp, sp2task, device_tasks, boys_table );

  }

template<ObaraSaikaType type_, int points_per_subtask_, int primpair_shared_limit_>
struct DeviceTask22 {
  static constexpr int max_primpair_shared_limit = 8;

  static constexpr int primpair_shared_limit = primpair_shared_limit_;
  static constexpr int points_per_subtask = points_per_subtask_;
  static constexpr int num_threads = points_per_subtask_;
  static constexpr ObaraSaikaType type = type_;

  static_assert(ObaraSaikaType::swap != type, "DeviceTask22 does not support swap");
  static constexpr bool diag = (ObaraSaikaType::diag == type);

  static constexpr bool use_shared = (primpair_shared_limit > 0) && 
                                     (primpair_shared_limit <= max_primpair_shared_limit);
  static constexpr int num_warps = points_per_subtask / cuda::warp_size;
  // Cannot declare shared memory array with length 0
  static constexpr int prim_buffer_size = (use_shared) ? num_warps * primpair_shared_limit : 1;

  using Params = ObaraSaikaParamsWithAB<type>;

  __inline__ __device__ static void compute( 
    const int i,
    const int npts,
    const int nprim_pairs,
    // Point data
    double4 (&s_task_data)[points_per_subtask],
    // Shell Pair Data
    const shell_pair* sp,
    // Output Data
    const Params param,
    int ldX,
    int ldG, 
    // Other
    double *boys_table) {

    // Unpack Params;
    const double *Xi = param.Xi;
    const double *Xj = param.Xj;
    double *Gi = param.Gi;
    double *Gj = param.Gj;
    const double X_AB = param.X_AB;
    const double Y_AB = param.Y_AB;
    const double Z_AB = param.Z_AB;

    const int laneId = threadIdx.x % cuda::warp_size;
    const int warpId __attribute__((unused)) = threadIdx.x / cuda::warp_size;

    const auto& prim_pairs = sp->prim_pairs();
    __shared__ GauXC::PrimitivePair<double> s_prim_pairs[prim_buffer_size] __attribute__((unused));

    if constexpr (use_shared) {
      load_primpair_shared(laneId, warpId, nprim_pairs,
        &(prim_pairs[0]), &(s_prim_pairs[warpId * primpair_shared_limit]));
      __syncwarp();
    }

    double outBuffer[6];
    double temp[num_threads * 31];

    // Loop over points in shared in batches of 32
    for (int i = 0; i <  num_warps; i++) {

      for (int j = 0; j < 6; j++) {
        outBuffer[j] = 0.0;
      }

      for(int j = 0; j < 31; ++j) SCALAR_STORE((temp + j), SCALAR_ZERO());

      const int pointIndex = i * cuda::warp_size + laneId;

      if (pointIndex < npts) {
        const double point_x = s_task_data[pointIndex].x;
        const double point_y = s_task_data[pointIndex].y;
        const double point_z = s_task_data[pointIndex].z;
        const double weight = s_task_data[pointIndex].w;

        for(int ij = 0; ij < nprim_pairs; ++ij) {
          const GauXC::PrimitivePair<double>* prim_pairs_use = nullptr; 
          if constexpr (use_shared) prim_pairs_use = &(s_prim_pairs[warpId * primpair_shared_limit]);
          else                      prim_pairs_use = &(prim_pairs[0]);

          double RHO = prim_pairs_use[ij].gamma;
          double RHO_INV = prim_pairs_use[ij].gamma_inv;
          double X_PA = prim_pairs_use[ij].PA.x;
          double Y_PA = prim_pairs_use[ij].PA.y;
          double Z_PA = prim_pairs_use[ij].PA.z;

          double xP = prim_pairs_use[ij].P.x;
          double yP = prim_pairs_use[ij].P.y;
          double zP = prim_pairs_use[ij].P.z;

          double eval = prim_pairs_use[ij].K_coeff_prod;

          // Evaluate T Values
          SCALAR_TYPE X_PC = SCALAR_SUB(xP, point_x);
          SCALAR_TYPE Y_PC = SCALAR_SUB(yP, point_y);
          SCALAR_TYPE Z_PC = SCALAR_SUB(zP, point_z);

          SCALAR_TYPE TVAL = SCALAR_MUL(X_PC, X_PC);
          TVAL = SCALAR_FMA(Y_PC, Y_PC, TVAL);
          TVAL = SCALAR_FMA(Z_PC, Z_PC, TVAL);
          TVAL = SCALAR_MUL(RHO, TVAL);

          SCALAR_TYPE t00, t01, t02, t03, t04, TVAL_inv_e;

          // Evaluate Boys function
          boys_element<4>(&TVAL, &TVAL_inv_e, &t04, boys_table);

          // Evaluate VRR Buffer
          SCALAR_TYPE t10, t11, t12, t13, t20, t21, t22, t30, t31, t40, tx, ty;

          t03 = SCALAR_MUL(SCALAR_ADD(SCALAR_MUL(TVAL, t04), TVAL_inv_e), SCALAR_SET1(0.28571428571428569843));
          t02 = SCALAR_MUL(SCALAR_ADD(SCALAR_MUL(TVAL, t03), TVAL_inv_e), SCALAR_SET1(0.40000000000000002220));
          t01 = SCALAR_MUL(SCALAR_ADD(SCALAR_MUL(TVAL, t02), TVAL_inv_e), SCALAR_SET1(0.66666666666666662966));
          t00 = SCALAR_MUL(SCALAR_ADD(SCALAR_MUL(TVAL, t01), TVAL_inv_e), SCALAR_SET1(2.00000000000000000000));

          t00 = SCALAR_MUL(eval, t00);
          t01 = SCALAR_MUL(eval, t01);
          t02 = SCALAR_MUL(eval, t02);
          t03 = SCALAR_MUL(eval, t03);
          t04 = SCALAR_MUL(eval, t04);
          t10 = SCALAR_MUL(X_PA, t00);
          t10 = SCALAR_FNMA(X_PC, t01, t10);
          t11 = SCALAR_MUL(X_PA, t01);
          t11 = SCALAR_FNMA(X_PC, t02, t11);
          t12 = SCALAR_MUL(X_PA, t02);
          t12 = SCALAR_FNMA(X_PC, t03, t12);
          t13 = SCALAR_MUL(X_PA, t03);
          t13 = SCALAR_FNMA(X_PC, t04, t13);
          t20 = SCALAR_MUL(X_PA, t10);
          t20 = SCALAR_FNMA(X_PC, t11, t20);
          tx = SCALAR_SUB(t00, t01);
          ty = SCALAR_SET1(0.5 * 1);
          ty = SCALAR_MUL(ty, RHO_INV);
          t20 = SCALAR_FMA(tx, ty, t20);
          t21 = SCALAR_MUL(X_PA, t11);
          t21 = SCALAR_FNMA(X_PC, t12, t21);
          tx = SCALAR_SUB(t01, t02);
          ty = SCALAR_SET1(0.5 * 1);
          ty = SCALAR_MUL(ty, RHO_INV);
          t21 = SCALAR_FMA(tx, ty, t21);
          t22 = SCALAR_MUL(X_PA, t12);
          t22 = SCALAR_FNMA(X_PC, t13, t22);
          tx = SCALAR_SUB(t02, t03);
          ty = SCALAR_SET1(0.5 * 1);
          ty = SCALAR_MUL(ty, RHO_INV);
          t22 = SCALAR_FMA(tx, ty, t22);
          tx = SCALAR_LOAD((temp + 0 ));
          tx = SCALAR_ADD(tx, t20);
          SCALAR_STORE((temp + 0 ), tx);
          t30 = SCALAR_MUL(X_PA, t20);
          t30 = SCALAR_FNMA(X_PC, t21, t30);
          tx = SCALAR_SUB(t10, t11);
          ty = SCALAR_SET1(0.5 * 2);
          ty = SCALAR_MUL(ty, RHO_INV);
          t30 = SCALAR_FMA(tx, ty, t30);
          t31 = SCALAR_MUL(X_PA, t21);
          t31 = SCALAR_FNMA(X_PC, t22, t31);
          tx = SCALAR_SUB(t11, t12);
          ty = SCALAR_SET1(0.5 * 2);
          ty = SCALAR_MUL(ty, RHO_INV);
          t31 = SCALAR_FMA(tx, ty, t31);
          tx = SCALAR_LOAD((temp + 6 ));
          tx = SCALAR_ADD(tx, t30);
          SCALAR_STORE((temp + 6 ), tx);
          t40 = SCALAR_MUL(X_PA, t30);
          t40 = SCALAR_FNMA(X_PC, t31, t40);
          tx = SCALAR_SUB(t20, t21);
          ty = SCALAR_SET1(0.5 * 3);
          ty = SCALAR_MUL(ty, RHO_INV);
          t40 = SCALAR_FMA(tx, ty, t40);
          tx = SCALAR_LOAD((temp + 16 ));
          tx = SCALAR_ADD(tx, t40);
          SCALAR_STORE((temp + 16 ), tx);
          t40 = SCALAR_MUL(Y_PA, t30);
          t40 = SCALAR_FNMA(Y_PC, t31, t40);
          tx = SCALAR_LOAD((temp + 17 ));
          tx = SCALAR_ADD(tx, t40);
          SCALAR_STORE((temp + 17 ), tx);
          t40 = SCALAR_MUL(Z_PA, t30);
          t40 = SCALAR_FNMA(Z_PC, t31, t40);
          tx = SCALAR_LOAD((temp + 18 ));
          tx = SCALAR_ADD(tx, t40);
          SCALAR_STORE((temp + 18 ), tx);
          t30 = SCALAR_MUL(Y_PA, t20);
          t30 = SCALAR_FNMA(Y_PC, t21, t30);
          t31 = SCALAR_MUL(Y_PA, t21);
          t31 = SCALAR_FNMA(Y_PC, t22, t31);
          tx = SCALAR_LOAD((temp + 7 ));
          tx = SCALAR_ADD(tx, t30);
          SCALAR_STORE((temp + 7 ), tx);
          t40 = SCALAR_MUL(Y_PA, t30);
          t40 = SCALAR_FNMA(Y_PC, t31, t40);
          tx = SCALAR_SUB(t20, t21);
          ty = SCALAR_SET1(0.5 * 1);
          ty = SCALAR_MUL(ty, RHO_INV);
          t40 = SCALAR_FMA(tx, ty, t40);
          tx = SCALAR_LOAD((temp + 19 ));
          tx = SCALAR_ADD(tx, t40);
          SCALAR_STORE((temp + 19 ), tx);
          t40 = SCALAR_MUL(Z_PA, t30);
          t40 = SCALAR_FNMA(Z_PC, t31, t40);
          tx = SCALAR_LOAD((temp + 20 ));
          tx = SCALAR_ADD(tx, t40);
          SCALAR_STORE((temp + 20 ), tx);
          t30 = SCALAR_MUL(Z_PA, t20);
          t30 = SCALAR_FNMA(Z_PC, t21, t30);
          t31 = SCALAR_MUL(Z_PA, t21);
          t31 = SCALAR_FNMA(Z_PC, t22, t31);
          tx = SCALAR_LOAD((temp + 8 ));
          tx = SCALAR_ADD(tx, t30);
          SCALAR_STORE((temp + 8 ), tx);
          t40 = SCALAR_MUL(Z_PA, t30);
          t40 = SCALAR_FNMA(Z_PC, t31, t40);
          tx = SCALAR_SUB(t20, t21);
          ty = SCALAR_SET1(0.5 * 1);
          ty = SCALAR_MUL(ty, RHO_INV);
          t40 = SCALAR_FMA(tx, ty, t40);
          tx = SCALAR_LOAD((temp + 21 ));
          tx = SCALAR_ADD(tx, t40);
          SCALAR_STORE((temp + 21 ), tx);
          t20 = SCALAR_MUL(Y_PA, t10);
          t20 = SCALAR_FNMA(Y_PC, t11, t20);
          t21 = SCALAR_MUL(Y_PA, t11);
          t21 = SCALAR_FNMA(Y_PC, t12, t21);
          t22 = SCALAR_MUL(Y_PA, t12);
          t22 = SCALAR_FNMA(Y_PC, t13, t22);
          tx = SCALAR_LOAD((temp + 1 ));
          tx = SCALAR_ADD(tx, t20);
          SCALAR_STORE((temp + 1 ), tx);
          t30 = SCALAR_MUL(Y_PA, t20);
          t30 = SCALAR_FNMA(Y_PC, t21, t30);
          tx = SCALAR_SUB(t10, t11);
          ty = SCALAR_SET1(0.5 * 1);
          ty = SCALAR_MUL(ty, RHO_INV);
          t30 = SCALAR_FMA(tx, ty, t30);
          t31 = SCALAR_MUL(Y_PA, t21);
          t31 = SCALAR_FNMA(Y_PC, t22, t31);
          tx = SCALAR_SUB(t11, t12);
          ty = SCALAR_SET1(0.5 * 1);
          ty = SCALAR_MUL(ty, RHO_INV);
          t31 = SCALAR_FMA(tx, ty, t31);
          tx = SCALAR_LOAD((temp + 9 ));
          tx = SCALAR_ADD(tx, t30);
          SCALAR_STORE((temp + 9 ), tx);
          t40 = SCALAR_MUL(Y_PA, t30);
          t40 = SCALAR_FNMA(Y_PC, t31, t40);
          tx = SCALAR_SUB(t20, t21);
          ty = SCALAR_SET1(0.5 * 2);
          ty = SCALAR_MUL(ty, RHO_INV);
          t40 = SCALAR_FMA(tx, ty, t40);
          tx = SCALAR_LOAD((temp + 22 ));
          tx = SCALAR_ADD(tx, t40);
          SCALAR_STORE((temp + 22 ), tx);
          t40 = SCALAR_MUL(Z_PA, t30);
          t40 = SCALAR_FNMA(Z_PC, t31, t40);
          tx = SCALAR_LOAD((temp + 23 ));
          tx = SCALAR_ADD(tx, t40);
          SCALAR_STORE((temp + 23 ), tx);
          t30 = SCALAR_MUL(Z_PA, t20);
          t30 = SCALAR_FNMA(Z_PC, t21, t30);
          t31 = SCALAR_MUL(Z_PA, t21);
          t31 = SCALAR_FNMA(Z_PC, t22, t31);
          tx = SCALAR_LOAD((temp + 10 ));
          tx = SCALAR_ADD(tx, t30);
          SCALAR_STORE((temp + 10 ), tx);
          t40 = SCALAR_MUL(Z_PA, t30);
          t40 = SCALAR_FNMA(Z_PC, t31, t40);
          tx = SCALAR_SUB(t20, t21);
          ty = SCALAR_SET1(0.5 * 1);
          ty = SCALAR_MUL(ty, RHO_INV);
          t40 = SCALAR_FMA(tx, ty, t40);
          tx = SCALAR_LOAD((temp + 24 ));
          tx = SCALAR_ADD(tx, t40);
          SCALAR_STORE((temp + 24 ), tx);
          t20 = SCALAR_MUL(Z_PA, t10);
          t20 = SCALAR_FNMA(Z_PC, t11, t20);
          t21 = SCALAR_MUL(Z_PA, t11);
          t21 = SCALAR_FNMA(Z_PC, t12, t21);
          t22 = SCALAR_MUL(Z_PA, t12);
          t22 = SCALAR_FNMA(Z_PC, t13, t22);
          tx = SCALAR_LOAD((temp + 2 ));
          tx = SCALAR_ADD(tx, t20);
          SCALAR_STORE((temp + 2 ), tx);
          t30 = SCALAR_MUL(Z_PA, t20);
          t30 = SCALAR_FNMA(Z_PC, t21, t30);
          tx = SCALAR_SUB(t10, t11);
          ty = SCALAR_SET1(0.5 * 1);
          ty = SCALAR_MUL(ty, RHO_INV);
          t30 = SCALAR_FMA(tx, ty, t30);
          t31 = SCALAR_MUL(Z_PA, t21);
          t31 = SCALAR_FNMA(Z_PC, t22, t31);
          tx = SCALAR_SUB(t11, t12);
          ty = SCALAR_SET1(0.5 * 1);
          ty = SCALAR_MUL(ty, RHO_INV);
          t31 = SCALAR_FMA(tx, ty, t31);
          tx = SCALAR_LOAD((temp + 11 ));
          tx = SCALAR_ADD(tx, t30);
          SCALAR_STORE((temp + 11 ), tx);
          t40 = SCALAR_MUL(Z_PA, t30);
          t40 = SCALAR_FNMA(Z_PC, t31, t40);
          tx = SCALAR_SUB(t20, t21);
          ty = SCALAR_SET1(0.5 * 2);
          ty = SCALAR_MUL(ty, RHO_INV);
          t40 = SCALAR_FMA(tx, ty, t40);
          tx = SCALAR_LOAD((temp + 25 ));
          tx = SCALAR_ADD(tx, t40);
          SCALAR_STORE((temp + 25 ), tx);
          t10 = SCALAR_MUL(Y_PA, t00);
          t10 = SCALAR_FNMA(Y_PC, t01, t10);
          t11 = SCALAR_MUL(Y_PA, t01);
          t11 = SCALAR_FNMA(Y_PC, t02, t11);
          t12 = SCALAR_MUL(Y_PA, t02);
          t12 = SCALAR_FNMA(Y_PC, t03, t12);
          t13 = SCALAR_MUL(Y_PA, t03);
          t13 = SCALAR_FNMA(Y_PC, t04, t13);
          t20 = SCALAR_MUL(Y_PA, t10);
          t20 = SCALAR_FNMA(Y_PC, t11, t20);
          tx = SCALAR_SUB(t00, t01);
          ty = SCALAR_SET1(0.5 * 1);
          ty = SCALAR_MUL(ty, RHO_INV);
          t20 = SCALAR_FMA(tx, ty, t20);
          t21 = SCALAR_MUL(Y_PA, t11);
          t21 = SCALAR_FNMA(Y_PC, t12, t21);
          tx = SCALAR_SUB(t01, t02);
          ty = SCALAR_SET1(0.5 * 1);
          ty = SCALAR_MUL(ty, RHO_INV);
          t21 = SCALAR_FMA(tx, ty, t21);
          t22 = SCALAR_MUL(Y_PA, t12);
          t22 = SCALAR_FNMA(Y_PC, t13, t22);
          tx = SCALAR_SUB(t02, t03);
          ty = SCALAR_SET1(0.5 * 1);
          ty = SCALAR_MUL(ty, RHO_INV);
          t22 = SCALAR_FMA(tx, ty, t22);
          tx = SCALAR_LOAD((temp + 3 ));
          tx = SCALAR_ADD(tx, t20);
          SCALAR_STORE((temp + 3 ), tx);
          t30 = SCALAR_MUL(Y_PA, t20);
          t30 = SCALAR_FNMA(Y_PC, t21, t30);
          tx = SCALAR_SUB(t10, t11);
          ty = SCALAR_SET1(0.5 * 2);
          ty = SCALAR_MUL(ty, RHO_INV);
          t30 = SCALAR_FMA(tx, ty, t30);
          t31 = SCALAR_MUL(Y_PA, t21);
          t31 = SCALAR_FNMA(Y_PC, t22, t31);
          tx = SCALAR_SUB(t11, t12);
          ty = SCALAR_SET1(0.5 * 2);
          ty = SCALAR_MUL(ty, RHO_INV);
          t31 = SCALAR_FMA(tx, ty, t31);
          tx = SCALAR_LOAD((temp + 12 ));
          tx = SCALAR_ADD(tx, t30);
          SCALAR_STORE((temp + 12 ), tx);
          t40 = SCALAR_MUL(Y_PA, t30);
          t40 = SCALAR_FNMA(Y_PC, t31, t40);
          tx = SCALAR_SUB(t20, t21);
          ty = SCALAR_SET1(0.5 * 3);
          ty = SCALAR_MUL(ty, RHO_INV);
          t40 = SCALAR_FMA(tx, ty, t40);
          tx = SCALAR_LOAD((temp + 26 ));
          tx = SCALAR_ADD(tx, t40);
          SCALAR_STORE((temp + 26 ), tx);
          t40 = SCALAR_MUL(Z_PA, t30);
          t40 = SCALAR_FNMA(Z_PC, t31, t40);
          tx = SCALAR_LOAD((temp + 27 ));
          tx = SCALAR_ADD(tx, t40);
          SCALAR_STORE((temp + 27 ), tx);
          t30 = SCALAR_MUL(Z_PA, t20);
          t30 = SCALAR_FNMA(Z_PC, t21, t30);
          t31 = SCALAR_MUL(Z_PA, t21);
          t31 = SCALAR_FNMA(Z_PC, t22, t31);
          tx = SCALAR_LOAD((temp + 13 ));
          tx = SCALAR_ADD(tx, t30);
          SCALAR_STORE((temp + 13 ), tx);
          t40 = SCALAR_MUL(Z_PA, t30);
          t40 = SCALAR_FNMA(Z_PC, t31, t40);
          tx = SCALAR_SUB(t20, t21);
          ty = SCALAR_SET1(0.5 * 1);
          ty = SCALAR_MUL(ty, RHO_INV);
          t40 = SCALAR_FMA(tx, ty, t40);
          tx = SCALAR_LOAD((temp + 28 ));
          tx = SCALAR_ADD(tx, t40);
          SCALAR_STORE((temp + 28 ), tx);
          t20 = SCALAR_MUL(Z_PA, t10);
          t20 = SCALAR_FNMA(Z_PC, t11, t20);
          t21 = SCALAR_MUL(Z_PA, t11);
          t21 = SCALAR_FNMA(Z_PC, t12, t21);
          t22 = SCALAR_MUL(Z_PA, t12);
          t22 = SCALAR_FNMA(Z_PC, t13, t22);
          tx = SCALAR_LOAD((temp + 4 ));
          tx = SCALAR_ADD(tx, t20);
          SCALAR_STORE((temp + 4 ), tx);
          t30 = SCALAR_MUL(Z_PA, t20);
          t30 = SCALAR_FNMA(Z_PC, t21, t30);
          tx = SCALAR_SUB(t10, t11);
          ty = SCALAR_SET1(0.5 * 1);
          ty = SCALAR_MUL(ty, RHO_INV);
          t30 = SCALAR_FMA(tx, ty, t30);
          t31 = SCALAR_MUL(Z_PA, t21);
          t31 = SCALAR_FNMA(Z_PC, t22, t31);
          tx = SCALAR_SUB(t11, t12);
          ty = SCALAR_SET1(0.5 * 1);
          ty = SCALAR_MUL(ty, RHO_INV);
          t31 = SCALAR_FMA(tx, ty, t31);
          tx = SCALAR_LOAD((temp + 14 ));
          tx = SCALAR_ADD(tx, t30);
          SCALAR_STORE((temp + 14 ), tx);
          t40 = SCALAR_MUL(Z_PA, t30);
          t40 = SCALAR_FNMA(Z_PC, t31, t40);
          tx = SCALAR_SUB(t20, t21);
          ty = SCALAR_SET1(0.5 * 2);
          ty = SCALAR_MUL(ty, RHO_INV);
          t40 = SCALAR_FMA(tx, ty, t40);
          tx = SCALAR_LOAD((temp + 29 ));
          tx = SCALAR_ADD(tx, t40);
          SCALAR_STORE((temp + 29 ), tx);
          t10 = SCALAR_MUL(Z_PA, t00);
          t10 = SCALAR_FNMA(Z_PC, t01, t10);
          t11 = SCALAR_MUL(Z_PA, t01);
          t11 = SCALAR_FNMA(Z_PC, t02, t11);
          t12 = SCALAR_MUL(Z_PA, t02);
          t12 = SCALAR_FNMA(Z_PC, t03, t12);
          t13 = SCALAR_MUL(Z_PA, t03);
          t13 = SCALAR_FNMA(Z_PC, t04, t13);
          t20 = SCALAR_MUL(Z_PA, t10);
          t20 = SCALAR_FNMA(Z_PC, t11, t20);
          tx = SCALAR_SUB(t00, t01);
          ty = SCALAR_SET1(0.5 * 1);
          ty = SCALAR_MUL(ty, RHO_INV);
          t20 = SCALAR_FMA(tx, ty, t20);
          t21 = SCALAR_MUL(Z_PA, t11);
          t21 = SCALAR_FNMA(Z_PC, t12, t21);
          tx = SCALAR_SUB(t01, t02);
          ty = SCALAR_SET1(0.5 * 1);
          ty = SCALAR_MUL(ty, RHO_INV);
          t21 = SCALAR_FMA(tx, ty, t21);
          t22 = SCALAR_MUL(Z_PA, t12);
          t22 = SCALAR_FNMA(Z_PC, t13, t22);
          tx = SCALAR_SUB(t02, t03);
          ty = SCALAR_SET1(0.5 * 1);
          ty = SCALAR_MUL(ty, RHO_INV);
          t22 = SCALAR_FMA(tx, ty, t22);
          tx = SCALAR_LOAD((temp + 5 ));
          tx = SCALAR_ADD(tx, t20);
          SCALAR_STORE((temp + 5 ), tx);
          t30 = SCALAR_MUL(Z_PA, t20);
          t30 = SCALAR_FNMA(Z_PC, t21, t30);
          tx = SCALAR_SUB(t10, t11);
          ty = SCALAR_SET1(0.5 * 2);
          ty = SCALAR_MUL(ty, RHO_INV);
          t30 = SCALAR_FMA(tx, ty, t30);
          t31 = SCALAR_MUL(Z_PA, t21);
          t31 = SCALAR_FNMA(Z_PC, t22, t31);
          tx = SCALAR_SUB(t11, t12);
          ty = SCALAR_SET1(0.5 * 2);
          ty = SCALAR_MUL(ty, RHO_INV);
          t31 = SCALAR_FMA(tx, ty, t31);
          tx = SCALAR_LOAD((temp + 15 ));
          tx = SCALAR_ADD(tx, t30);
          SCALAR_STORE((temp + 15 ), tx);
          t40 = SCALAR_MUL(Z_PA, t30);
          t40 = SCALAR_FNMA(Z_PC, t31, t40);
          tx = SCALAR_SUB(t20, t21);
          ty = SCALAR_SET1(0.5 * 3);
          ty = SCALAR_MUL(ty, RHO_INV);
          t40 = SCALAR_FMA(tx, ty, t40);
          tx = SCALAR_LOAD((temp + 30 ));
          tx = SCALAR_ADD(tx, t40);
          SCALAR_STORE((temp + 30 ), tx);
        }

        bool nonzero = false;
        for(int i = 0; i < 31; ++i) {
          nonzero = nonzero || abs(temp[i ]) > 1e-12;
        }

        if (diag || nonzero) {
          const double * __restrict__ Xik = (Xi + pointIndex);
          const double * __restrict__ Xjk = (Xj + pointIndex);
          double * __restrict__ Gik = (Gi + pointIndex);
          double * __restrict__ Gjk = (Gj + pointIndex);

          SCALAR_TYPE const_value_v = weight;

          double const_value, X_ABp, Y_ABp, Z_ABp, comb_m_i, comb_n_j, comb_p_k;
          SCALAR_TYPE const_value_w;
          SCALAR_TYPE tx, ty, tz, tw, t0, t1, t2, t3, t4, t5;

          X_ABp = 1.0; comb_m_i = 1.0;
          Y_ABp = 1.0; comb_n_j = 1.0;
          Z_ABp = 1.0; comb_p_k = 1.0;
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          ty = SCALAR_LOAD((Xjk + 0 * ldX));
          t0 = SCALAR_LOAD((temp + 16 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_MUL(tx, t0);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 17 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 18 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 19 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 20 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 21 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
                                  
          X_ABp = SCALAR_MUL(X_ABp, X_AB); comb_m_i = SCALAR_MUL(comb_m_i * 2, SCALAR_RECIPROCAL(1));
          Y_ABp = 1.0; comb_n_j = 1.0;
          Z_ABp = 1.0; comb_p_k = 1.0;
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          t0 = SCALAR_LOAD((temp + 6 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_FMA(tx, t0, tw);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 7 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 8 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 9 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 10 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 11 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
                                  
          X_ABp = SCALAR_MUL(X_ABp, X_AB); comb_m_i = SCALAR_MUL(comb_m_i * 1, SCALAR_RECIPROCAL(2));
          Y_ABp = 1.0; comb_n_j = 1.0;
          Z_ABp = 1.0; comb_p_k = 1.0;
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          t0 = SCALAR_LOAD((temp + 0 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_FMA(tx, t0, tw);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 1 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 2 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 3 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 4 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 5 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
          if constexpr (!diag) atomicAdd((Gjk + 0 * ldG), tw);
    


          X_ABp = 1.0; comb_m_i = 1.0;
          Y_ABp = 1.0; comb_n_j = 1.0;
          Z_ABp = 1.0; comb_p_k = 1.0;
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          ty = SCALAR_LOAD((Xjk + 1 * ldX));
          t0 = SCALAR_LOAD((temp + 17 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_MUL(tx, t0);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 19 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 20 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 22 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 23 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 24 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
                                  
          Y_ABp = SCALAR_MUL(Y_ABp, Y_AB); comb_n_j = SCALAR_MUL(comb_n_j * 1, SCALAR_RECIPROCAL(1));
          Z_ABp = 1.0; comb_p_k = 1.0;
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          t0 = SCALAR_LOAD((temp + 6 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_FMA(tx, t0, tw);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 7 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 8 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 9 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 10 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 11 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
                                  
          X_ABp = SCALAR_MUL(X_ABp, X_AB); comb_m_i = SCALAR_MUL(comb_m_i * 1, SCALAR_RECIPROCAL(1));
          Y_ABp = 1.0; comb_n_j = 1.0;
          Z_ABp = 1.0; comb_p_k = 1.0;
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          t0 = SCALAR_LOAD((temp + 7 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_FMA(tx, t0, tw);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 9 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 10 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 12 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 13 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 14 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
                                  
          Y_ABp = SCALAR_MUL(Y_ABp, Y_AB); comb_n_j = SCALAR_MUL(comb_n_j * 1, SCALAR_RECIPROCAL(1));
          Z_ABp = 1.0; comb_p_k = 1.0;
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          t0 = SCALAR_LOAD((temp + 0 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_FMA(tx, t0, tw);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 1 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 2 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 3 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 4 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 5 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
          if constexpr (!diag) atomicAdd((Gjk + 1 * ldG), tw);




          X_ABp = 1.0; comb_m_i = 1.0;
          Y_ABp = 1.0; comb_n_j = 1.0;
          Z_ABp = 1.0; comb_p_k = 1.0;
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          ty = SCALAR_LOAD((Xjk + 2 * ldX));
          t0 = SCALAR_LOAD((temp + 18 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_MUL(tx, t0);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 20 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 21 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 23 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 24 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 25 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
                                  
          Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 1, SCALAR_RECIPROCAL(1));
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          t0 = SCALAR_LOAD((temp + 6 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_FMA(tx, t0, tw);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 7 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 8 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 9 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 10 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 11 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
                                  
          X_ABp = SCALAR_MUL(X_ABp, X_AB); comb_m_i = SCALAR_MUL(comb_m_i * 1, SCALAR_RECIPROCAL(1));
          Y_ABp = 1.0; comb_n_j = 1.0;
          Z_ABp = 1.0; comb_p_k = 1.0;
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          t0 = SCALAR_LOAD((temp + 8 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_FMA(tx, t0, tw);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 10 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 11 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 13 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 14 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 15 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
                                  
          Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 1, SCALAR_RECIPROCAL(1));
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          t0 = SCALAR_LOAD((temp + 0 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_FMA(tx, t0, tw);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 1 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 2 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 3 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 4 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 5 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
          if constexpr (!diag) atomicAdd((Gjk + 2 * ldG), tw);





          X_ABp = 1.0; comb_m_i = 1.0;
          Y_ABp = 1.0; comb_n_j = 1.0;
          Z_ABp = 1.0; comb_p_k = 1.0;
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          ty = SCALAR_LOAD((Xjk + 3 * ldX));
          t0 = SCALAR_LOAD((temp + 19 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_MUL(tx, t0);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 22 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 23 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 26 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 27 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 28 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
                                  
          Y_ABp = SCALAR_MUL(Y_ABp, Y_AB); comb_n_j = SCALAR_MUL(comb_n_j * 2, SCALAR_RECIPROCAL(1));
          Z_ABp = 1.0; comb_p_k = 1.0;
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          t0 = SCALAR_LOAD((temp + 7 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_FMA(tx, t0, tw);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 9 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 10 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 12 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 13 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 14 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
                                  
          Y_ABp = SCALAR_MUL(Y_ABp, Y_AB); comb_n_j = SCALAR_MUL(comb_n_j * 1, SCALAR_RECIPROCAL(2));
          Z_ABp = 1.0; comb_p_k = 1.0;
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          t0 = SCALAR_LOAD((temp + 0 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_FMA(tx, t0, tw);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 1 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 2 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 3 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 4 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 5 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
          if constexpr (!diag) atomicAdd((Gjk + 3 * ldG), tw);



          X_ABp = 1.0; comb_m_i = 1.0;
          Y_ABp = 1.0; comb_n_j = 1.0;
          Z_ABp = 1.0; comb_p_k = 1.0;
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          ty = SCALAR_LOAD((Xjk + 4 * ldX));
          t0 = SCALAR_LOAD((temp + 20 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_MUL(tx, t0);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 23 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 24 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 27 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 28 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 29 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
                                  
          Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 1, SCALAR_RECIPROCAL(1));
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          t0 = SCALAR_LOAD((temp + 7 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_FMA(tx, t0, tw);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 9 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 10 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 12 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 13 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 14 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
                                  
          Y_ABp = SCALAR_MUL(Y_ABp, Y_AB); comb_n_j = SCALAR_MUL(comb_n_j * 1, SCALAR_RECIPROCAL(1));
          Z_ABp = 1.0; comb_p_k = 1.0;
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          t0 = SCALAR_LOAD((temp + 8 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_FMA(tx, t0, tw);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 10 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 11 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 13 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 14 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 15 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
                                  
          Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 1, SCALAR_RECIPROCAL(1));
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          t0 = SCALAR_LOAD((temp + 0 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_FMA(tx, t0, tw);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 1 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 2 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 3 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 4 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 5 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
          if constexpr (!diag) atomicAdd((Gjk + 4 * ldG), tw);




          X_ABp = 1.0; comb_m_i = 1.0;
          Y_ABp = 1.0; comb_n_j = 1.0;
          Z_ABp = 1.0; comb_p_k = 1.0;
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          ty = SCALAR_LOAD((Xjk + 5 * ldX));
          t0 = SCALAR_LOAD((temp + 21 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_MUL(tx, t0);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 24 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 25 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 28 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 29 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 30 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
                                  
          Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 2, SCALAR_RECIPROCAL(1));
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          t0 = SCALAR_LOAD((temp + 8 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_FMA(tx, t0, tw);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 10 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 11 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 13 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 14 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 15 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
                                  
          Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 1, SCALAR_RECIPROCAL(2));
          const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
          const_value_w = SCALAR_MUL(const_value_v, const_value);
          tx = SCALAR_LOAD((Xik + 0 * ldX));
          t0 = SCALAR_LOAD((temp + 0 ));
          t0 = SCALAR_MUL(t0, const_value_w);
          tz = SCALAR_MUL(ty, t0);
          tw = SCALAR_FMA(tx, t0, tw);
          //atomicAdd((Gik + 0 * ldG), tz);
          outBuffer[0] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 1 * ldX));
          t1 = SCALAR_LOAD((temp + 1 ));
          t1 = SCALAR_MUL(t1, const_value_w);
          tz = SCALAR_MUL(ty, t1);
          tw = SCALAR_FMA(tx, t1, tw);
          //atomicAdd((Gik + 1 * ldG), tz);
          outBuffer[1] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 2 * ldX));
          t2 = SCALAR_LOAD((temp + 2 ));
          t2 = SCALAR_MUL(t2, const_value_w);
          tz = SCALAR_MUL(ty, t2);
          tw = SCALAR_FMA(tx, t2, tw);
          //atomicAdd((Gik + 2 * ldG), tz);
          outBuffer[2] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 3 * ldX));
          t3 = SCALAR_LOAD((temp + 3 ));
          t3 = SCALAR_MUL(t3, const_value_w);
          tz = SCALAR_MUL(ty, t3);
          tw = SCALAR_FMA(tx, t3, tw);
          //atomicAdd((Gik + 3 * ldG), tz);
          outBuffer[3] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 4 * ldX));
          t4 = SCALAR_LOAD((temp + 4 ));
          t4 = SCALAR_MUL(t4, const_value_w);
          tz = SCALAR_MUL(ty, t4);
          tw = SCALAR_FMA(tx, t4, tw);
          //atomicAdd((Gik + 4 * ldG), tz);
          outBuffer[4] += tz;
                                  
          tx = SCALAR_LOAD((Xik + 5 * ldX));
          t5 = SCALAR_LOAD((temp + 5 ));
          t5 = SCALAR_MUL(t5, const_value_w);
          tz = SCALAR_MUL(ty, t5);
          tw = SCALAR_FMA(tx, t5, tw);
          //atomicAdd((Gik + 5 * ldG), tz);
          outBuffer[5] += tz;
          if constexpr (!diag) atomicAdd((Gjk + 5 * ldG), tw);

          atomicAdd((Gik + 0 * ldG), outBuffer[0]);
          atomicAdd((Gik + 1 * ldG), outBuffer[1]);
          atomicAdd((Gik + 2 * ldG), outBuffer[2]);
          atomicAdd((Gik + 3 * ldG), outBuffer[3]);
          atomicAdd((Gik + 4 * ldG), outBuffer[4]);
          atomicAdd((Gik + 5 * ldG), outBuffer[5]);
        }
      }
    }
    __syncwarp();
  }
};

template <int primpair_limit>
using AM22 = DeviceTask22<ObaraSaikaType::base,
  alg_constants::CudaAoSScheme1::ObaraSaika::points_per_subtask, primpair_limit>;

template <int primpair_limit>
using AM2 = DeviceTask22<ObaraSaikaType::diag,
  alg_constants::CudaAoSScheme1::ObaraSaika::points_per_subtask, primpair_limit>;

  void integral_2_2_task_batched(
    size_t ntasks, size_t nsubtask,
    int max_primpair, size_t max_nsp,
    GauXC::XCDeviceTask*                device_tasks,
    const GauXC::TaskToShellPairDevice* task2sp,
    const std::array<int32_t, 4>*  subtasks,
    const int32_t* nprim_pairs_device,
    shell_pair** sp_ptr_device,
    double* sp_X_AB_device,
    double* sp_Y_AB_device,
    double* sp_Z_AB_device,
    double *boys_table,
    hipStream_t stream) {

    int nblocks_x = nsubtask;
    int nblocks_y = 8; 
    int nblocks_z = 1;
    dim3 nblocks(nblocks_x, nblocks_y, nblocks_z);
    dim3 nthreads(alg_constants::CudaAoSScheme1::ObaraSaika::points_per_subtask);
    
    dev_integral_task_map_dispatcher<AM22>(
      nblocks, nthreads, max_primpair, stream, 
      ntasks, nsubtask,
      device_tasks, task2sp, 
      (int4*) subtasks, nprim_pairs_device, sp_ptr_device,
      sp_X_AB_device, sp_Y_AB_device, sp_Z_AB_device,
      boys_table );
  }

  void integral_2_task_batched(
    size_t ntasks, size_t nsubtask,
    int max_primpair, size_t max_nsp,
    GauXC::XCDeviceTask*                device_tasks,
    const GauXC::TaskToShellPairDevice* task2sp,
    const std::array<int32_t, 4>*  subtasks,
    const int32_t* nprim_pairs_device,
    shell_pair** sp_ptr_device,
    double* sp_X_AB_device,
    double* sp_Y_AB_device,
    double* sp_Z_AB_device,
    double *boys_table,
    hipStream_t stream) {

    int nblocks_x = nsubtask;
    int nblocks_y = 8; 
    int nblocks_z = 1;
    dim3 nblocks(nblocks_x, nblocks_y, nblocks_z);
    dim3 nthreads(alg_constants::CudaAoSScheme1::ObaraSaika::points_per_subtask);
    
    dev_integral_task_map_dispatcher<AM2>(
      nblocks, nthreads, max_primpair, stream, 
      ntasks, nsubtask,
      device_tasks, task2sp, 
      (int4*) subtasks, nprim_pairs_device, sp_ptr_device,
      sp_X_AB_device, sp_Y_AB_device, sp_Z_AB_device,
      boys_table );
  }

}
